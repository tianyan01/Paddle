#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
// #define DEBUG_MOE_TMPROFILE
#include "paddle/fluid/operators/fused/fused_multi_transformer_moe_op.h"
#include "paddle/phi/kernels/funcs/scatter.cu.h"
#ifdef DEBUG_MOE_TMPROFILE
#include "paddle/fluid/platform/timer.h"
#endif
#if defined(PADDLE_WITH_CUTLASS)
#include "paddle/phi/kernels/fusion/cutlass/cutlass_kernels/moe_gemm/moe_gemm_kernels_template.h"
#endif
DECLARE_bool(enable_moe_gemm_cutlass);
namespace paddle {
namespace operators {

using Tensor = phi::DenseTensor;
// #define _DEBUG_FUSED_MULTI_TRANSFORMER
inline bool CheckFlashAttn(const phi::GPUContext &dev_ctx,
                           const phi::DenseTensor &x) {
  int dev = dev_ctx.GetPlace().GetDeviceId();
  if (!paddle::platform::IsSupportFlashAttn(dev)) {
    return false;
  }
  return (x.dtype() == DataType::FLOAT16);
}
template <typename T>
class FusedMultiTransformerMoeOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    auto &dev_ctx = ctx.cuda_device_context();
#ifdef DEBUG_MOE_TMPROFILE
    platform::Timer all_tm, other_tm, trans_tm;
    platform::Timer qkv_tm, fmha_tm, out_linear_tm;
    platform::Timer expert_tm, ln_tm, gate_tm;
    platform::Timer gate_nccl_tm, gather_tm, scatter_tm;
    all_tm.Start();
    other_tm.Start();
#endif
    auto *time_step = ctx.Input<Tensor>("TimeStep");
    // 0. input
    auto *input_x = ctx.Input<Tensor>("X");
    const auto input_x_dims = input_x->dims();
    int bsz = input_x_dims[0];
    int seq_len = input_x_dims[1];
    int dim_embed = input_x_dims[2];
    int bsz_seq = bsz * seq_len;
    if (bsz_seq == 0) {
      return;
    }
    // LOG(INFO) << "intput X: bsz: " << bsz << ", seq_len: " << seq_len << ",
    // dim_embed: " << dim_embed;
    const std::string act_method = ctx.Attr<std::string>("act_method");
    auto *sequence_lengths = ctx.Input<Tensor>("SeqLengths");  // nullptr
    auto *beam_cache_offset = ctx.Input<Tensor>("BeamCacheOffset");
    int beam_size = 1;
    if (beam_cache_offset) {
      beam_size = beam_cache_offset->dims()[1];
    }
    // LOG(INFO) << "beam_size: " << beam_size;

    auto *out = ctx.Output<Tensor>("Out");
    dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));

    // 1. layer norm
    const auto pre_layer_norm = ctx.Attr<bool>("pre_layer_norm");
    if (!pre_layer_norm) {
      VLOG(0) << "not support post layer norm!";
      return;
    }
    const float epsilon = ctx.Attr<float>("epsilon");
    auto ln_scales = ctx.MultiInput<Tensor>("LnScale");
    auto ln_biases = ctx.MultiInput<Tensor>("LnBias");

    auto ln_compute = AttnLayerNorm<T>(dev_ctx, epsilon, bsz_seq, dim_embed);
    Tensor ln_mean, ln_var;
    ln_mean.Resize({{bsz_seq}});
    auto *ln_mean_data =
        dev_ctx.Alloc<U>(&ln_mean, ln_mean.numel() * sizeof(U));
    ln_var.Resize({{bsz_seq}});
    auto *ln_var_data = dev_ctx.Alloc<U>(&ln_var, ln_var.numel() * sizeof(U));

    // 2. qkv
    // x: qkv's input [batch_size, seq_len, dim_embed]
    // y: qkv's weight: [3, num_head, dim_head, dim_embed]
    auto qkv_weights = ctx.MultiInput<Tensor>("QKVW");
    auto qkv_biases = ctx.MultiInput<Tensor>("QKVBias");
    const bool trans_qkvw = ctx.Attr<bool>("trans_qkvw");
    const auto qkv_w_dims = qkv_weights[0]->dims();
    int num_head = trans_qkvw ? qkv_w_dims[1] : qkv_w_dims[2];
    int dim_head = trans_qkvw ? qkv_w_dims[2] : qkv_w_dims[3];
    int hidden_size = num_head * dim_head;
    int output_size = 3 * hidden_size;
    int input_size = dim_embed;

    bool compute_bias = qkv_biases.size() > 0 && time_step == nullptr;
    // (transA, transB, compute_bias) = (false, trans_qkvw, false)
    auto qkv_compute = AttnMatMul<T>(dev_ctx,
                                     false,
                                     trans_qkvw,
                                     bsz_seq,
                                     output_size,
                                     input_size,
                                     compute_bias);
#if defined(PADDLE_WITH_CUTLASS)
    using InputType = typename phi::PDDataTypeTraits<T>::DataType;
    phi::MoeGemmRunner<InputType, InputType> gemm_runner;
    auto default_act = phi::getActivationType("none");
    auto expert_act = phi::getActivationType(act_method);
#else
    PADDLE_ENFORCE_EQ(FLAGS_enable_moe_gemm_cutlass, false, 
                      "not support cutlass fused moe gemm please disable "
                      "FLAGS_enable_moe_gemm_cutlass");
#endif
    Tensor qkv_out;
    qkv_out.Resize({{bsz, seq_len, 3, num_head, dim_head}});
    auto *qkv_out_data =
        dev_ctx.Alloc<T>(&qkv_out, qkv_out.numel() * sizeof(T));

    // 3. fmha
    auto dropout_implementation =
        ctx.Attr<std::string>("dropout_implementation");
    AttnDropoutParam attn_param(
        true, dropout_implementation, 0.0, true, true, 0, nullptr);
    auto fmha_compute =
        FMHARef<T>(dev_ctx, bsz, seq_len, num_head, dim_head, attn_param);

    // check support flash attn
    bool is_support_flash_attn = CheckFlashAttn(dev_ctx, *input_x);
    auto fmha_fa_compute = FlashAttnFMHARef<plat::float16>(
        dev_ctx, bsz, seq_len, num_head, dim_head, attn_param);
    auto *src_mask = ctx.Input<Tensor>("SrcMask");
    auto cache_kvs = ctx.MultiInput<Tensor>("CacheKV");
    auto cache_kv_outs = ctx.MultiOutput<Tensor>("CacheKVOut");

    int time_step_cpu = 0;
    if (time_step) {
      time_step_cpu = src_mask->dims()[3] - 1;
    }

    auto out_seq_len = seq_len;
    if (time_step) {
      PADDLE_ENFORCE_GT(
          time_step_cpu,
          0,
          platform::errors::PreconditionNotMet(
              "The value of time_step must > 0, but now is %d", time_step_cpu));
      PADDLE_ENFORCE_EQ(
          seq_len,
          1,
          platform::errors::PreconditionNotMet(
              "In decode stage, the seq_len of input must be 1, but now is %d",
              seq_len));
      out_seq_len += time_step_cpu;
    }

    Tensor transpose_out_2, qk_out;
    transpose_out_2.Resize({{3, bsz, num_head, seq_len, dim_head}});
    auto *transpose_out_2_data =
        dev_ctx.Alloc<T>(&transpose_out_2, transpose_out_2.numel() * sizeof(T));

    Tensor softmax_out;
    Tensor attn_dropout_mask_out, attn_dropout_out;
    Tensor qktv_out, fmha_out;
    if (!is_support_flash_attn) {
      qk_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
      auto *qk_out_data = dev_ctx.Alloc<T>(&qk_out, qk_out.numel() * sizeof(T));

      softmax_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
      auto *softmax_out_data =
          dev_ctx.Alloc<T>(&softmax_out, softmax_out.numel() * sizeof(T));
      qktv_out.Resize({{bsz, num_head, seq_len, dim_head}});
      auto *qktv_out_data =
          dev_ctx.Alloc<T>(&qktv_out, qktv_out.numel() * sizeof(T));
    }

    fmha_out.Resize({{bsz, seq_len, num_head, dim_head}});
    auto *fmha_out_data =
        dev_ctx.Alloc<T>(&fmha_out, fmha_out.numel() * sizeof(T));

    // 4. out_linear
    auto out_linear_weights = ctx.MultiInput<Tensor>("OutLinearW");
    auto out_linear_biases = ctx.MultiInput<Tensor>("OutLinearBias");
    int ring_id = ctx.Attr<int>("ring_id");
    // (transA, transB, compute_bias) = (false, false, false)
    auto out_linear_compute = AttnMatMul<T>(
        dev_ctx, false, false, bsz_seq, dim_embed, hidden_size, false);

    // 5. ln(residual + bias), pre layernorm in ffn/moe
    DropoutParam dropout_param(false, 0, true, true, 0.0, nullptr, 0);
    FusedDropoutLayerNormHelper<T, uint8_t> fused_dropout_layernorm_helper(
        dev_ctx, bsz_seq, dim_embed, dropout_param, epsilon);
    auto ffn_ln_scales = ctx.MultiInput<Tensor>("FFNLnScale");
    auto ffn_ln_biases = ctx.MultiInput<Tensor>("FFNLnBias");
    Tensor bias_dropout_residual_out, dropout_mask_out;
    T *bias_dropout_residual_out_data = nullptr;
    bias_dropout_residual_out.Resize({{bsz_seq, dim_embed}});
    bias_dropout_residual_out_data =
        dev_ctx.Alloc<T>(&bias_dropout_residual_out,
                         bias_dropout_residual_out.numel() * sizeof(T));
    uint8_t *dropout_mask_out_data = nullptr;

    // 6. moe layer: gate / expert_w & b / some attrs
    auto gate_weights = ctx.MultiInput<Tensor>("GateWeight");
    auto gate_biases = ctx.MultiInput<Tensor>("GateBias");
    auto expert_weights1 = ctx.MultiInput<Tensor>("ExpertWeight1");
    auto expert_biases1 = ctx.MultiInput<Tensor>("ExpertBias1");
    auto expert_weights2 = ctx.MultiInput<Tensor>("ExpertWeight2");
    auto expert_biases2 = ctx.MultiInput<Tensor>("ExpertBias2");
    int dim_feedforward = expert_weights1[0]->dims()[1];
    // gemm cutlass used ColumnMajor store
    if (FLAGS_enable_moe_gemm_cutlass) {
      dim_feedforward = expert_weights1[0]->dims()[0];  // batched gemm
    }
    int topk = ctx.Attr<int>("topk");
    int mp_size = ctx.Attr<int>("mp_size");
    int mp_rank = ctx.Attr<int>("mp_rank");
    int num_expert = ctx.Attr<int>("num_expert");
    int world_size = ctx.Attr<int>("world_size");
    int moe_ring_id = ctx.Attr<int>("moe_ring_id");
    bool approximate = ctx.Attr<bool>("approximate");

    int tot_expert = world_size * num_expert;
    // after slice, bsz_seq should be change
    int sliced_bsz_seq = bsz_seq;
    int start = 0;
    int end = 0;
    if (mp_size > 1) {
      start = bsz_seq / world_size * mp_rank;
      end = std::min(start + bsz_seq / world_size, bsz_seq);
      sliced_bsz_seq = end - start;
    }
    int out_batch_size = sliced_bsz_seq * topk;
    // slice
    Tensor sliced_inp;
    sliced_inp.Resize({{sliced_bsz_seq, dim_embed}});
    dev_ctx.Alloc<T>(&sliced_inp, sliced_inp.numel() * sizeof(T));
    // gate linear
    Tensor gate_out;
    gate_out.Resize({{sliced_bsz_seq, tot_expert}});
    dev_ctx.Alloc<T>(&gate_out, gate_out.numel() * sizeof(T));
    // topk
    Tensor topk_value, topk_idx;
    topk_value.Resize({{sliced_bsz_seq, topk}});
    dev_ctx.Alloc<T>(&topk_value, topk_value.numel() * sizeof(T));
    topk_idx.Resize({{sliced_bsz_seq, topk}});
    dev_ctx.Alloc<int64_t>(&topk_idx, topk_idx.numel() * sizeof(int64_t));
    // local expert count, global expert count
    Tensor local_expert_count, global_expert_count;
    local_expert_count.Resize({{tot_expert}});
    global_expert_count.Resize({{tot_expert}});
    dev_ctx.Alloc<int64_t>(&local_expert_count,
                           local_expert_count.numel() * sizeof(int64_t));
    dev_ctx.Alloc<int64_t>(&global_expert_count,
                           global_expert_count.numel() * sizeof(int64_t));

    // fwd_expert_count, fwd_batch_size
    Tensor fwd_expert_count, fwd_expert_csum_len;
    Tensor fwd_expert_csum_len_cpu;
    fwd_expert_count.Resize({{num_expert}});
    fwd_expert_csum_len.Resize({{num_expert + 1}});
    dev_ctx.Alloc<int64_t>(&fwd_expert_count,
                           fwd_expert_count.numel() * sizeof(int64_t));
    dev_ctx.Alloc<int64_t>(&fwd_expert_csum_len,
                           fwd_expert_csum_len.numel() * sizeof(int64_t));
    phi::funcs::set_constant<int64_t>(
        dev_ctx, &fwd_expert_csum_len, static_cast<int64_t>(0));

    // pos, temp pos
    Tensor pos, temp_pos;
    pos.Resize({{out_batch_size}});
    temp_pos.Resize({{out_batch_size}});
    dev_ctx.Alloc<int64_t>(&pos, pos.numel() * sizeof(int64_t));
    dev_ctx.Alloc<int64_t>(&temp_pos, temp_pos.numel() * sizeof(int64_t));
    // cumsum
    Tensor lec_cum;
    lec_cum.Resize({{tot_expert}});
    dev_ctx.Alloc<int64_t>(&lec_cum, lec_cum.numel() * sizeof(int64_t));
    // fused moe ffn tmp out
    Tensor index_select_out;
    index_select_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&index_select_out, index_select_out.numel() * sizeof(T));
    Tensor global_gather_out;
    global_gather_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&global_gather_out, global_gather_out.numel() * sizeof(T));
    Tensor moe_gather_out;
    moe_gather_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&moe_gather_out, moe_gather_out.numel() * sizeof(T));
    Tensor bmm_out;
    bmm_out.Resize({{sliced_bsz_seq, 1, dim_embed}});
    dev_ctx.Alloc<T>(&bmm_out, bmm_out.numel() * sizeof(T));
    Tensor all_gather_out;
    all_gather_out.Resize({{bsz_seq, dim_embed}});
    dev_ctx.Alloc<T>(&all_gather_out, all_gather_out.numel() * sizeof(T));
    // topk tensor
    Tensor topk_tensor;
    topk_tensor.Resize({{1}});
    dev_ctx.Alloc<int64_t>(&topk_tensor, topk_tensor.numel() * sizeof(int64_t));
    phi::FullKernel<int64_t, phi::GPUContext>(
        dev_ctx, {1}, topk, pos.dtype(), &topk_tensor);
    // moe nccl
    phi::NCCLMoECollective moe_pg(dev_ctx, moe_ring_id, num_expert);

    Tensor buf0, moe_out;
    buf0.Resize({{bsz_seq, dim_embed}});
    dev_ctx.Alloc<T>(&buf0, buf0.numel() * sizeof(T));
    moe_out.ShareDataWith(*out);
    moe_out.Resize({{bsz_seq, dim_embed}});

    const T *x_data = input_x->data<T>();
#ifdef DEBUG_MOE_TMPROFILE
    dev_ctx.Wait();
    other_tm.Pause();
#endif
    int layers = qkv_weights.size();
    for (int i = 0; i < layers; ++i) {
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step1, pre layernorm";
#endif
#ifdef DEBUG_MOE_TMPROFILE
      trans_tm.Resume();
#endif
      // step1. layer_norm, only layer 0
      if (i == 0) {
#ifdef DEBUG_MOE_TMPROFILE
        ln_tm.Resume();
#endif
        auto *ln_scale_data = ln_scales[i]->data<U>();
        auto *ln_bias_data = ln_biases[i]->data<U>();
        // TODO(wangxi): can remove mean var in inference
        ln_compute.ComputeForward(x_data,
                                  ln_scale_data,
                                  ln_bias_data,
                                  buf0.data<T>(),
                                  ln_mean_data,
                                  ln_var_data);
#ifdef DEBUG_MOE_TMPROFILE
        dev_ctx.Wait();
        ln_tm.Pause();
#endif
      }
      // step2. qkv
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step2, qkv";
#endif
      const Tensor *qkv_bias = qkv_biases.size() > 0 ? qkv_biases[i] : nullptr;
      // NOTE: in decoder stage, bias is fused in fmha
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      qkv_tm.Resume();
#endif
      const Tensor *bias = time_step ? nullptr : qkv_bias;
      qkv_compute.ComputeForward(
          qkv_weights[i], &buf0, bias, &qkv_out, &qkv_out);
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      qkv_tm.Pause();
      fmha_tm.Resume();
#endif
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step3.1 fmha";
#endif
      // step3. fmha
      const Tensor *cache_kv = cache_kvs.size() > 0 ? cache_kvs[i] : nullptr;
      Tensor *cache_kv_out = cache_kv ? cache_kv_outs[i] : nullptr;

      if (time_step) {  // generation decoder stage
        // [2, batch_size, num_head, max_seq_len, head_size]
        int max_seq_len = cache_kv->dims()[3];
        fmha<T>(dev_ctx,
                qkv_out,
                *qkv_bias,
                *src_mask,
                sequence_lengths,
                nullptr,
                beam_cache_offset,
                cache_kv_out,
                &fmha_out,
                bsz,
                beam_size,
                max_seq_len,
                num_head,
                dim_head,
                time_step_cpu,
                0,
                1. / sqrt(dim_head));
      } else if (cache_kv_out) {  // generation encoder stage
        if (is_support_flash_attn) {
          fmha_fa_compute.ComputeForward(qkv_out,
                                         nullptr,
                                         src_mask,
                                         &transpose_out_2,
                                         nullptr,
                                         &softmax_out,  // softmax_lse_out
                                         &attn_dropout_mask_out,  // seek_offset
                                         &attn_dropout_out,       // softmax_out
                                         &fmha_out);
          // input: [bs, seq_len, 3, num_head, head_dim]
          // output: [3, bs, num_head, seq_len, head_dim]
          std::vector<int> perm_1 = {2, 0, 3, 1, 4};
          transpose_out_2.Resize({{3, bsz, num_head, seq_len, dim_head}});
          TransposeGPUKernelDriver<T>(
              dev_ctx, qkv_out, perm_1, &transpose_out_2);
        } else {
          fmha_compute.ComputeForward(qkv_out,
                                      nullptr,
                                      src_mask,
                                      &transpose_out_2,
                                      nullptr,
                                      &qk_out,
                                      nullptr,
                                      &softmax_out,
                                      &attn_dropout_mask_out,
                                      &attn_dropout_out,
                                      &qktv_out,
                                      &fmha_out);
        }
        // [3, bsz, num_head, seq_len, head_dim]
        T *qkv_data = transpose_out_2_data;
        int64_t q_size = bsz * seq_len * num_head * dim_head;
        int64_t k_size = q_size;
        const T *q_ptr = qkv_data;
        const T *k_ptr = q_ptr + q_size;
        const T *v_ptr = k_ptr + k_size;

        // [2, bsz, num_head, max_seq_len, head_dim]
        int max_seq_len = cache_kv_out->dims()[3];
        T *cache_kv_data = cache_kv_out->data<T>();
        int64_t cache_k_size = bsz * num_head * max_seq_len * dim_head;

        T *cache_k_ptr = cache_kv_data;
        T *cache_v_ptr = cache_kv_data + cache_k_size;

        write_cache_kv<T>(dev_ctx,
                          cache_k_ptr,
                          cache_v_ptr,
                          k_ptr,
                          v_ptr,
                          bsz,
                          num_head,
                          seq_len,
                          max_seq_len,
                          dim_head);
      } else {  // not generation
        VLOG(0) << "not support!";
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step3.2 out linear";
#endif
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      fmha_tm.Pause();
      out_linear_tm.Resume();
#endif
      // 输出到buf0
      out_linear_compute.ComputeForward(
          out_linear_weights[i], &fmha_out, nullptr, &buf0, nullptr);
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      out_linear_tm.Pause();
#endif
      if (mp_size > 1) {
        phi::AllReduce<T>(buf0, ring_id, buf0.numel(), dev_ctx);
      }

#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step4";
#endif

      // step5. ln(residual + dropout(input + bias))，在MHA里的
      auto *ln_scale_data = ffn_ln_scales[i]->data<U>();
      auto *ln_bias_data = ffn_ln_biases[i]->data<U>();
      auto *out_linear_bias_data = out_linear_biases[i]->data<T>();
#ifdef DEBUG_MOE_TMPROFILE
      ln_tm.Resume();
#endif
      // pre layer norm : bias_dropout_residual_out is residual
      fused_dropout_layernorm_helper.LayernormResidualDropoutBias(
          dev_ctx,
          buf0.data<T>(),
          x_data,  // residual, moe out
          out_linear_bias_data,
          ln_scale_data,
          ln_bias_data,
          bias_dropout_residual_out_data,
          dropout_mask_out_data,
          buf0.data<T>(),  // output to buf0
          ln_mean_data,
          ln_var_data);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step5";
#endif
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      ln_tm.Pause();
#endif
      // moe
      // step2 resize and slice ln_out
      if (mp_size > 1) {
        sliced_inp = buf0.Slice(start, end);
      } else {
        sliced_inp = buf0;
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, gate & topk";
#endif
#ifdef DEBUG_MOE_TMPROFILE
      gate_tm.Resume();
#endif
      // step3 gate & topk
      phi::MatMulAndAdd<T>(dev_ctx,
                           gate_weights[i],
                           &sliced_inp,
                           gate_biases[i],
                           false,
                           false,
                           true,  //  compute bias
                           &gate_out,
                           &gate_out);
      phi::TopkKernel<T, phi::GPUContext>(dev_ctx,
                                          gate_out,
                                          topk,  // scalar
                                          -1,
                                          true,
                                          false,
                                          &topk_value,
                                          &topk_idx);
      // step4 prepare forward
      // step4.1 number count
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, number count";
#endif
      phi::NumberCountKernel<int64_t, phi::GPUContext>(
          dev_ctx, topk_idx, tot_expert, &local_expert_count);
      // step4.2 all_to_all
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, all_to_all";
#endif
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      gate_nccl_tm.Resume();
#endif
      if (world_size > 1) {
        moe_pg.AllToAll<int64_t>(local_expert_count, global_expert_count);
      } else {
        global_expert_count = local_expert_count;
      }
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      gate_nccl_tm.Pause();
#endif
      // global expert count resize
      global_expert_count.Resize({{world_size, num_expert}});
      // fwd expert count
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, fwd expert count";
#endif
      phi::SumKernel<int64_t, phi::GPUContext>(dev_ctx,
                                               global_expert_count,
                                               phi::IntArray({0}),
                                               global_expert_count.dtype(),
                                               false,
                                               &fwd_expert_count);
      // fwd batch size
      phi::CumsumTensorValue<int64_t>(
          dev_ctx, fwd_expert_count, &fwd_expert_csum_len, 1);
      // step4.3 cumsum & assign pos
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, cumsum";
#endif
      phi::CumsumTensorValue<int64_t>(dev_ctx, local_expert_count, &lec_cum);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, assign pos";
#endif
      phi::AssignInsAndPosCompute<int64_t>(
          dev_ctx, &lec_cum, &topk_idx, &pos, out_batch_size, topk, &temp_pos);

#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, tensor copy";
#endif
      framework::TensorCopy(
          fwd_expert_csum_len, platform::CPUPlace(), &fwd_expert_csum_len_cpu);
      dev_ctx.Wait();
      int fwd_bsz = fwd_expert_csum_len_cpu.data<int64_t>()[num_expert];

      Tensor global_scatter_out;
      global_scatter_out.Resize({{fwd_bsz, dim_embed}});
      dev_ctx.Alloc<T>(&global_scatter_out,
                       global_scatter_out.numel() * sizeof(T));

      Tensor all_expert_out;
      all_expert_out.Resize({{fwd_bsz, dim_embed}});
      dev_ctx.Alloc<T>(&all_expert_out, all_expert_out.numel() * sizeof(T));

      // step 5, MOEScatter
      // step 5.1, index select
      // suppose tmp_pos->shape != [0]
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, index select";
#endif
      phi::IndexSelectKernel<T, phi::GPUContext>(
          dev_ctx, sliced_inp, temp_pos, 0, &index_select_out);
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      gate_tm.Pause();

      dev_ctx.Wait();
      scatter_tm.Resume();
#endif
      if (world_size > 1) {
        moe_pg.Scatter<T>(&index_select_out,
                          local_expert_count,
                          global_expert_count,
                          &global_scatter_out);
      } else {
        global_scatter_out = index_select_out;
      }
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      scatter_tm.Pause();

      dev_ctx.Wait();
      expert_tm.Resume();
#endif
      // step 6, Expert Computation
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, Expert Computation";
#endif
      if (fwd_bsz != 0) {
        // encoder, use matmul
        Tensor expert_out1;
        if (FLAGS_enable_moe_gemm_cutlass) {
#if defined(PADDLE_WITH_CUTLASS)
          int expert_idx = i * num_expert;
          // csum length
          int64_t *total_rows_before_expert =
              fwd_expert_csum_len.data<int64_t>();
          const T *permuted_data = global_scatter_out.data<T>();
          const T *fc1_expert_weights = expert_weights1[expert_idx]->data<T>();
          const T *fc_scales = nullptr;
          const T *fc1_expert_biases = expert_biases1[expert_idx]->data<T>();

#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
          std::ostringstream ostr;
          int64_t *pnum = fwd_expert_csum_len_cpu.data<int64_t>();
          for (int j = 0; j <= num_expert; ++j) {
            ostr << pnum[j] << ",";
          }
          VLOG(0)
              << "layer id=" << i << ", expert_idx=" << expert_idx
              << ", numel=" << fwd_expert_count.numel()
              << ", dim_feedforward=" << dim_feedforward
              << ", dim_embed=" << dim_embed << ", num_expert=" << num_expert
              << ", global_scatter_out=" << global_scatter_out.dims()
              << ", expert_weights1=" << expert_weights1[expert_idx]->dims()
              << ", start ptr="
              << (int64_t)(expert_weights1[expert_idx]->data()) << ", end ptr="
              << (int64_t)(expert_weights1[expert_idx + num_expert - 1]->data())
              << ", numel=" << expert_weights1[expert_idx]->numel()
              << ", expert_weights2=" << expert_weights2[expert_idx]->dims()
              << ", expert nums=" << ostr.str();
#endif

          expert_out1.Resize({{ fwd_bsz, dim_feedforward }});
          dev_ctx.Alloc<T>(&expert_out1, expert_out1.numel() * sizeof(T));

          T *fc1_result = expert_out1.data<T>();

          gemm_runner.moe_gemm_bias_act(
              reinterpret_cast<const InputType *>(permuted_data),
              reinterpret_cast<const InputType *>(fc1_expert_weights),
              reinterpret_cast<const InputType *>(fc_scales),
              reinterpret_cast<const InputType *>(fc1_expert_biases),
              reinterpret_cast<InputType *>(fc1_result),
              total_rows_before_expert,
              fwd_bsz,
              dim_feedforward,
              dim_embed,
              num_expert,
              static_cast<phi::ActivationType>(expert_act),
              dev_ctx.stream());

          const T *fc2_expert_weights = expert_weights2[expert_idx]->data<T>();
          const T *fc2_expert_biases = expert_biases2[expert_idx]->data<T>();
          T *fc2_result = all_expert_out.data<T>();

          gemm_runner.moe_gemm_bias_act(
              reinterpret_cast<const InputType *>(fc1_result),
              reinterpret_cast<const InputType *>(fc2_expert_weights),
              reinterpret_cast<const InputType *>(fc_scales),
              reinterpret_cast<const InputType *>(fc2_expert_biases),
              reinterpret_cast<InputType *>(fc2_result),
              total_rows_before_expert,
              fwd_bsz,
              dim_embed,
              dim_feedforward,
              num_expert,
              static_cast<phi::ActivationType>(default_act),
              dev_ctx.stream());
#endif
        } else {
          int last_index = 0;
          int64_t *csum_len = fwd_expert_csum_len_cpu.data<int64_t>();
          for (int idx = 0; idx < num_expert; idx++) {
            int end = csum_len[idx + 1];
            int cur_expert_count = end - last_index;
            if (cur_expert_count <= 0) {
              continue;
            }

            expert_out1.Resize({{cur_expert_count, dim_feedforward}});
            dev_ctx.Alloc<T>(&expert_out1, expert_out1.numel() * sizeof(T));

            Tensor tmp_inp = global_scatter_out.Slice(last_index, end);
            int expert_idx = i * num_expert + idx;
            // cuda 11.4
#if (CUDA_VERSION >= 11040)
            phi::MatMulAndAddGelu<T>(dev_ctx,
                                     expert_weights1[expert_idx],
                                     &tmp_inp,
                                     expert_biases1[expert_idx],
                                     false,
                                     false,
                                     false,  // dont compute bias
                                     &expert_out1);
#else
            // linear1 matmul
            // VLOG(0) << "moe, Expert Computation, linear1 mul";
            phi::MatMulAndAdd<T>(dev_ctx,
                                 expert_weights1[expert_idx],
                                 &tmp_inp,
                                 nullptr,
                                 false,
                                 false,
                                 false,  // dont compute bias
                                 &expert_out1,
                                 nullptr);
            // bias gelu
            FusedDropoutHelper<T, uint8_t> fused_act_dropout_helper(
                dev_ctx, cur_expert_count, dim_feedforward, dropout_param);
            // VLOG(0) << "moe, Expert Computation, add bias & gelu";
            // inplace
            fused_act_dropout_helper.DropoutActBias(
                dev_ctx,
                expert_out1.data<T>(),
                expert_biases1[expert_idx]->data<T>(),
                "gelu",
                expert_out1.data<T>(),
                nullptr,
                1.0,
                nullptr,
                0,
                1.0,
                1,
                127.0,
                -127.0,
                approximate);
#endif
            // linear2 matmul & add
            // VLOG(0) << "moe, Expert Computation, linear2 matmul & add";
            Tensor expert_out2 = all_expert_out.Slice(last_index, end);
            phi::MatMulAndAdd<T>(dev_ctx,
                                 expert_weights2[expert_idx],
                                 &expert_out1,
                                 expert_biases2[expert_idx],
                                 false,
                                 false,
                                 true,  //  compute bias
                                 &expert_out2,
                                 &expert_out2);
            last_index = end;
          }
        }
        // at last, concat all expert out
      } else {
        all_expert_out = global_scatter_out;
      }
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      expert_tm.Pause();
      gather_tm.Resume();
#endif
      // step7. MOEGather
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, MOEGather";
#endif
      if (world_size > 1) {
        moe_pg.Gather<T>(&all_expert_out, &global_gather_out);
      } else {
        global_gather_out = all_expert_out;
      }
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      gather_tm.Pause();
#endif
      // step 7.2, local_gather or scatter
      // suppose pos->shape != [0]
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, local_gather or scatter";
#endif
      phi::funcs::GPUScatterAssign<T, int64_t>(
          dev_ctx, global_gather_out, pos, &moe_gather_out, true);

      // step 8, reshape & bmm
      // moe gather out reshape
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, reshape & bmm";
#endif
      moe_gather_out.Resize({{sliced_bsz_seq, topk, dim_embed}});
      topk_value.Resize({{sliced_bsz_seq, 1, topk}});
      phi::BmmKernel<T, phi::GPUContext>(
          dev_ctx, topk_value, moe_gather_out, &bmm_out);
      bmm_out.Resize({{sliced_bsz_seq, dim_embed}});
      // step 9, AllGather
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, AllGather";
#endif
      if (mp_size > 1) {
        // all gather
        moe_pg.AllGather<T>(bmm_out, all_gather_out);
      } else {
        all_gather_out = bmm_out;
      }
      // step 11, add residual
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, add residual";
#endif
      if (i < layers - 1) {
        // add residual & next layer norm
#ifdef DEBUG_MOE_TMPROFILE
        dev_ctx.Wait();
        ln_tm.Resume();
#endif
        auto *ln_scale_data = ln_scales[i + 1]->data<U>();
        auto *ln_bias_data = ln_biases[i + 1]->data<U>();
        fused_dropout_layernorm_helper.LayernormResidualDropoutBias(
            dev_ctx,
            all_gather_out.data<T>(),        // src
            bias_dropout_residual_out_data,  // residual
            nullptr,                         // bias
            ln_scale_data,
            ln_bias_data,
            moe_out.data<T>(),  // add out, next layer real input, for residual
            dropout_mask_out_data,
            buf0.data<T>(),  // out, after layernorm
            ln_mean_data,
            ln_var_data);
#ifdef DEBUG_MOE_TMPROFILE
        dev_ctx.Wait();
        ln_tm.Pause();
#endif
      } else {
        // last layer, only add residual
        phi::AddKernel<T, phi::GPUContext>(
            dev_ctx, all_gather_out, bias_dropout_residual_out, &moe_out);
      }
      x_data = moe_out.data<T>();
#ifdef DEBUG_MOE_TMPROFILE
      dev_ctx.Wait();
      trans_tm.Pause();
#endif
    }  // layers loop end
    out->Resize({{bsz, seq_len, dim_embed}});
#ifdef DEBUG_MOE_TMPROFILE
    dev_ctx.Wait();
    all_tm.Pause();
    VLOG(0) << "gpu=" << static_cast<int>(dev_ctx.GetPlace().GetDeviceId())
            << ", bsz=" << bsz << ", seq_len=" << seq_len
            << ", total span=" << all_tm.ElapsedMS()
            << ", input=" << other_tm.ElapsedMS()
            << ", transformer=" << trans_tm.ElapsedMS()
            << ", [qkv=" << qkv_tm.ElapsedMS()
            << ", fmha=" << fmha_tm.ElapsedMS()
            << ", out_linear=" << out_linear_tm.ElapsedMS()
            << ", expert=" << expert_tm.ElapsedMS()
            << ", ln=" << ln_tm.ElapsedMS()
            << ", gate/all2all=" << gate_tm.ElapsedMS() << "/"
            << gate_nccl_tm.ElapsedMS()
            << ", scatter=" << scatter_tm.ElapsedMS()
            << ", gather=" << gather_tm.ElapsedMS() << "]";
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(fused_multi_transformer_moe,
                        ops::FusedMultiTransformerMoeOpKernel<plat::float16>,
                        ops::FusedMultiTransformerMoeOpKernel<float>);
