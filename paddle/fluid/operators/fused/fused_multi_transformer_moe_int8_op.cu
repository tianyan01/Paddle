#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
// #define DEBUG_MOE_TMPROFILE_INT8
#include "paddle/fluid/operators/fused/fused_multi_transformer_moe_op.h"
#include "paddle/fluid/operators/fused/layernorm_quant_dequant.h"
#include "paddle/phi/kernels/funcs/scatter.cu.h"
#ifdef DEBUG_MOE_TMPROFILE_INT8
#include "paddle/fluid/platform/timer.h"
#endif
namespace paddle {
namespace operators {

using Tensor = phi::DenseTensor;
// #define _DEBUG_FUSED_MULTI_TRANSFORMER

template <typename T>
static void PrintMatrix(const T *mat_d, int num, std::string name) {
  std::vector<T> tmp(num);
  hipMemcpy(tmp.data(), mat_d, sizeof(T) * num, hipMemcpyDeviceToHost);

  std::ofstream outfile;
  outfile.open(name + ".txt", std::ios::out);
  std::stringstream ss;

  for (int i = 0; i < num; ++i) {
    if (std::is_same<T, int8_t>::value) {
      ss << static_cast<int>(tmp[i]) << std::endl;
    } else {
      ss << std::setprecision(8) << tmp[i] << std::endl;
    }
  }
  outfile << ss.str();
  outfile.close();
}
// #define _DEBUG_FUSED_MULTI_TRANSFORMER
inline bool CheckFlashAttn(const phi::GPUContext &dev_ctx,
                           const phi::DenseTensor &x) {
  int dev = dev_ctx.GetPlace().GetDeviceId();
  if (!paddle::platform::IsSupportFlashAttn(dev)) {
    return false;
  }
  return (x.dtype() == DataType::FLOAT16);
}
template <typename T>
class FusedMultiTransformerMoeINT8OpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    auto &dev_ctx = ctx.cuda_device_context();
#ifdef DEBUG_MOE_TMPROFILE_INT8
    platform::Timer all_tm, other_tm, trans_tm;
    platform::Timer qkv_tm, fmha_tm, out_linear_tm;
    platform::Timer expert_tm, ln_tm, gate_tm;
    platform::Timer gate_nccl_tm, gather_tm, scatter_tm;
    all_tm.Start();
    other_tm.Start();
#endif
    auto *time_step = ctx.Input<Tensor>("TimeStep");
    // 0. input
    auto *input_x = ctx.Input<Tensor>("X");
    const auto input_x_dims = input_x->dims();
    int bsz = input_x_dims[0];
    int seq_len = input_x_dims[1];
    int dim_embed = input_x_dims[2];
    int bsz_seq = bsz * seq_len;
    if (bsz_seq == 0) {
      return;
    }

    // quant input scales, vector, size = num_layers
    auto qkv_in_scale = ctx.Attr<std::vector<float>>("qkv_in_scale");
    auto out_linear_in_scale =
        ctx.Attr<std::vector<float>>("out_linear_in_scale");
    // moe expert scales, vector, size = num_expert * num_layers
    auto expert_weight1_in_scale =
        ctx.Attr<std::vector<float>>("expert_weight1_in_scale");
    auto expert_weight2_in_scale =
        ctx.Attr<std::vector<float>>("expert_weight2_in_scale");

    // quant round type and bound
    auto quant_round_type = ctx.Attr<int>("quant_round_type");
    auto quant_max_bound = ctx.Attr<float>("quant_max_bound");
    auto quant_min_bound = ctx.Attr<float>("quant_min_bound");

    // dequant output scales, vertor<tensor>, size = [num_layers, n], n is gemm
    // output size
    auto qkv_out_scales = ctx.MultiInput<Tensor>("QKVOutScale");
    auto out_linear_out_scales = ctx.MultiInput<Tensor>("OutLinearOutScale");
    // dequant output scales, tensor, size = [num_layers * num_expert, n], n is
    // gemm output size
    auto expert_weight1_out_scales =
        ctx.MultiInput<Tensor>("ExpertWeight1OutScale");
    auto expert_weight2_out_scales =
        ctx.MultiInput<Tensor>("ExpertWeight2OutScale");

    auto *sequence_lengths = ctx.Input<Tensor>("SeqLengths");
    auto *beam_cache_offset = ctx.Input<Tensor>("BeamCacheOffset");
    int beam_size = 1;
    if (beam_cache_offset) {
      beam_size = beam_cache_offset->dims()[1];
    }

    // 1. layer norm
    const auto pre_layer_norm = ctx.Attr<bool>("pre_layer_norm");
    if (!pre_layer_norm) {
      VLOG(0) << "not support post layer norm!";
      return;
    }
    const float epsilon = ctx.Attr<float>("epsilon");
    auto ln_scales = ctx.MultiInput<Tensor>("LnScale");
    auto ln_biases = ctx.MultiInput<Tensor>("LnBias");

    // in type is T, out type is int8_t
    auto ln_compute =
        AttnLayerNorm<T, T, int8_t>(dev_ctx, epsilon, bsz_seq, dim_embed);
    Tensor ln_mean, ln_var;
    ln_mean.Resize({{bsz_seq}});
    auto *ln_mean_data =
        dev_ctx.Alloc<U>(&ln_mean, ln_mean.numel() * sizeof(U));
    ln_var.Resize({{bsz_seq}});
    auto *ln_var_data = dev_ctx.Alloc<U>(&ln_var, ln_var.numel() * sizeof(U));

    // 2. qkv
    // x: qkv's input [batch_size, seq_len, dim_embed]
    // y: qkv's weight: [3, num_head, dim_head, dim_embed]
    auto qkv_weights = ctx.MultiInput<Tensor>("QKVW");
    auto qkv_biases = ctx.MultiInput<Tensor>("QKVBias");
    const bool trans_qkvw = ctx.Attr<bool>("trans_qkvw");
    const auto qkv_w_dims = qkv_weights[0]->dims();
    int num_head = trans_qkvw ? qkv_w_dims[1] : qkv_w_dims[2];
    int dim_head = trans_qkvw ? qkv_w_dims[2] : qkv_w_dims[3];
    int hidden_size = num_head * dim_head;
    int output_size = 3 * hidden_size;
    int input_size = dim_embed;

    bool compute_bias = qkv_biases.size() > 0 && time_step == nullptr;
    // (transA, transB, compute_bias) = (false, trans_qkvw, false)
    AttnMatmulINT8<T> qkv_compute(
        dev_ctx, bsz_seq, output_size, input_size, compute_bias);
    Tensor qkv_out;
    qkv_out.Resize({{bsz, seq_len, 3, num_head, dim_head}});
    auto *qkv_out_data =
        dev_ctx.Alloc<T>(&qkv_out, qkv_out.numel() * sizeof(T));

    // 3. fmha
    // check support flash attn
    AttnDropoutParam attn_param(
        true, "upscale_in_train", 0.0, true, true, 0, nullptr);
    bool is_support_flash_attn = CheckFlashAttn(dev_ctx, *input_x);
    auto fmha_fa_compute = FlashAttnFMHARef<plat::float16>(
        dev_ctx, bsz, seq_len, num_head, dim_head, attn_param);
    auto fmha_compute =
        FMHARef<T>(dev_ctx, bsz, seq_len, num_head, dim_head, attn_param);
    auto *src_mask = ctx.Input<Tensor>("SrcMask");
    auto cache_kvs = ctx.MultiInput<Tensor>("CacheKV");
    auto cache_kv_outs = ctx.MultiOutput<Tensor>("CacheKVOut");

    int time_step_cpu = 0;
    auto out_seq_len = seq_len;
    if (time_step) {
      time_step_cpu = src_mask->dims()[3] - 1;
      out_seq_len += time_step_cpu;
    }

    Tensor transpose_out_2, qk_out;
    transpose_out_2.Resize({{3, bsz, num_head, seq_len, dim_head}});
    auto *transpose_out_2_data =
        dev_ctx.Alloc<T>(&transpose_out_2, transpose_out_2.numel() * sizeof(T));

    Tensor softmax_out;
    Tensor attn_dropout_mask_out, attn_dropout_out;
    Tensor qktv_out, fmha_out;
    if (!is_support_flash_attn) {
      qk_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
      auto *qk_out_data = dev_ctx.Alloc<T>(&qk_out, qk_out.numel() * sizeof(T));
      softmax_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
      auto *softmax_out_data =
          dev_ctx.Alloc<T>(&softmax_out, softmax_out.numel() * sizeof(T));

      qktv_out.Resize({{bsz, num_head, seq_len, dim_head}});
      auto *qktv_out_data =
          dev_ctx.Alloc<T>(&qktv_out, qktv_out.numel() * sizeof(T));
    }
    fmha_out.Resize({{bsz, seq_len, num_head, dim_head}});
    auto *fmha_out_data =
        dev_ctx.Alloc<T>(&fmha_out, fmha_out.numel() * sizeof(T));

    // 4. out_linear  注意！！这个weight的维度需要transpose！！
    auto out_linear_weights = ctx.MultiInput<Tensor>("OutLinearW");
    auto out_linear_biases = ctx.MultiInput<Tensor>("OutLinearBias");
    int ring_id = ctx.Attr<int>("ring_id");
    // (transA, transB, compute_bias) = (false, false, false)
    AttnMatmulINT8<T> out_linear_compute(
        dev_ctx, bsz_seq, dim_embed, hidden_size, false);

    // 5. ln(residual + bias)
    DropoutParam dropout_param(false, 0, true, true, 0.0, nullptr, 0);

    using LayerNormComputeType = float;
    auto ffn_ln_scales = ctx.MultiInput<Tensor>("FFNLnScale");
    auto ffn_ln_biases = ctx.MultiInput<Tensor>("FFNLnBias");
    Tensor bias_dropout_residual_out, dropout_mask_out;
    T *bias_dropout_residual_out_data = nullptr;
    bias_dropout_residual_out.Resize({{bsz_seq, dim_embed}});
    bias_dropout_residual_out_data =
        dev_ctx.Alloc<T>(&bias_dropout_residual_out,
                         bias_dropout_residual_out.numel() * sizeof(T));
    uint8_t *dropout_mask_out_data = nullptr;

    // 6. moe layer: gate / expert_w & b / some attrs
    auto gate_weights = ctx.MultiInput<Tensor>("GateWeight");
    auto gate_biases = ctx.MultiInput<Tensor>("GateBias");
    // weight的维度需要transpose！！！！
    auto expert_weights1 = ctx.MultiInput<Tensor>("ExpertWeight1");
    auto expert_biases1 = ctx.MultiInput<Tensor>("ExpertBias1");
    auto expert_weights2 = ctx.MultiInput<Tensor>("ExpertWeight2");
    auto expert_biases2 = ctx.MultiInput<Tensor>("ExpertBias2");
    int dim_feedforward =
        expert_weights1[0]->dims()[0];  // dim is [dim_feedforward, dim_embed]
    int topk = ctx.Attr<int>("topk");
    int mp_size = ctx.Attr<int>("mp_size");
    int mp_rank = ctx.Attr<int>("mp_rank");
    int num_expert = ctx.Attr<int>("num_expert");
    int world_size = ctx.Attr<int>("world_size");
    int moe_ring_id = ctx.Attr<int>("moe_ring_id");
    bool approximate = ctx.Attr<bool>("approximate");

    int tot_expert = world_size * num_expert;
    // after slice, bsz_seq should be change
    int sliced_bsz_seq = bsz_seq;
    int start = 0;
    int end = 0;
    if (mp_size > 1) {
      start = bsz_seq / world_size * mp_rank;
      end = std::min(start + bsz_seq / world_size, bsz_seq);
      sliced_bsz_seq = end - start;
    }
    int out_batch_size = sliced_bsz_seq * topk;
    // slice
    Tensor sliced_inp;
    sliced_inp.Resize({{sliced_bsz_seq, dim_embed}});
    dev_ctx.Alloc<T>(&sliced_inp, sliced_inp.numel() * sizeof(T));
    // gate linear
    Tensor gate_out;
    gate_out.Resize({{sliced_bsz_seq, tot_expert}});
    dev_ctx.Alloc<T>(&gate_out, gate_out.numel() * sizeof(T));
    // topk
    Tensor topk_value, topk_idx;
    topk_value.Resize({{sliced_bsz_seq, topk}});
    dev_ctx.Alloc<T>(&topk_value, topk_value.numel() * sizeof(T));
    topk_idx.Resize({{sliced_bsz_seq, topk}});
    dev_ctx.Alloc<int64_t>(&topk_idx, topk_idx.numel() * sizeof(int64_t));
    // local expert count, global expert count
    Tensor local_expert_count, global_expert_count;
    local_expert_count.Resize({{tot_expert}});
    global_expert_count.Resize({{tot_expert}});
    dev_ctx.Alloc<int64_t>(&local_expert_count,
                           local_expert_count.numel() * sizeof(int64_t));
    dev_ctx.Alloc<int64_t>(&global_expert_count,
                           global_expert_count.numel() * sizeof(int64_t));
    // fwd_expert_count, fwd_batch_size
    Tensor fwd_expert_count, fwd_batch_size;
    Tensor fwd_expert_count_cpu, fwd_batch_size_cpu;
    fwd_expert_count.Resize({{num_expert}});
    fwd_batch_size.Resize({{1}});
    dev_ctx.Alloc<int64_t>(&fwd_expert_count,
                           fwd_expert_count.numel() * sizeof(int64_t));
    dev_ctx.Alloc<int64_t>(&fwd_batch_size,
                           fwd_batch_size.numel() * sizeof(int64_t));
    // pos, temp pos
    Tensor pos, temp_pos;
    pos.Resize({{out_batch_size}});
    temp_pos.Resize({{out_batch_size}});
    dev_ctx.Alloc<int64_t>(&pos, pos.numel() * sizeof(int64_t));
    dev_ctx.Alloc<int64_t>(&temp_pos, temp_pos.numel() * sizeof(int64_t));
    // cumsum
    Tensor lec_cum;
    lec_cum.Resize({{tot_expert}});
    dev_ctx.Alloc<int64_t>(&lec_cum, lec_cum.numel() * sizeof(int64_t));
    // fused moe ffn tmp out
    Tensor index_select_out;
    index_select_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&index_select_out, index_select_out.numel() * sizeof(T));
    Tensor global_gather_out;
    global_gather_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&global_gather_out, global_gather_out.numel() * sizeof(T));
    Tensor moe_gather_out;
    moe_gather_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&moe_gather_out, moe_gather_out.numel() * sizeof(T));
    Tensor bmm_out;
    bmm_out.Resize({{sliced_bsz_seq, 1, dim_embed}});
    dev_ctx.Alloc<T>(&bmm_out, bmm_out.numel() * sizeof(T));
    Tensor all_gather_out;
    all_gather_out.Resize({{bsz_seq, dim_embed}});
    dev_ctx.Alloc<T>(&all_gather_out, all_gather_out.numel() * sizeof(T));
    // topk tensor
    Tensor topk_tensor;
    topk_tensor.Resize({{1}});
    dev_ctx.Alloc<int64_t>(&topk_tensor, topk_tensor.numel() * sizeof(int64_t));
    phi::FullKernel<int64_t, phi::GPUContext>(
        dev_ctx, {1}, topk, pos.dtype(), &topk_tensor);

    // moe nccl
    phi::NCCLMoECollective moe_pg(dev_ctx, moe_ring_id, num_expert);

    // []. init workspace for cublasLt transform
    Tensor input_workspace, output_workspace, cublaslt_workspace;
    // for input and output transform data is CUBLASLT_ORDER_COL32 format,
    int m_max = bsz_seq, k_max = std::max({dim_embed, dim_feedforward}),
        n_max = std::max({output_size, dim_embed, dim_feedforward});
    // maybe need to change the size of workspace here

    input_workspace.Resize({{(m_max * k_max + 31) / 32 * 32}});
    dev_ctx.Alloc<int8_t>(&input_workspace,
                          input_workspace.numel() * sizeof(int8_t));

    output_workspace.Resize({{(n_max * m_max + 31) / 32 * 32}});
    dev_ctx.Alloc<int32_t>(&output_workspace,
                           output_workspace.numel() * sizeof(int32_t));

    cublaslt_workspace.Resize({{3000000}});
    dev_ctx.Alloc<int8_t>(&cublaslt_workspace,
                          cublaslt_workspace.numel() * sizeof(int8_t));

    // calc
    auto *out = ctx.Output<Tensor>("Out");
    auto *from_data = dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));

    Tensor buf0, moe_out;
    buf0.Resize({{bsz_seq, dim_embed}});
    dev_ctx.Alloc<T>(&buf0, buf0.numel() * sizeof(T));
    moe_out.ShareDataWith(*out);
    moe_out.Resize({{bsz_seq, dim_embed}});

    const T *x_data = input_x->data<T>();
#ifdef DEBUG_MOE_TMPROFILE_INT8
    dev_ctx.Wait();
    other_tm.Pause();
#endif
    int layers = qkv_weights.size();
    for (int i = 0; i < layers; ++i) {
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step1, pre layernorm";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      trans_tm.Resume();
#endif
      // step1. layer_norm
      if (i == 0) {
#ifdef DEBUG_MOE_TMPROFILE_INT8
        ln_tm.Resume();
#endif
        auto *ln_scale_data = ln_scales[i]->data<U>();
        auto *ln_bias_data = ln_biases[i]->data<U>();
        // layer norm后，对输出做scale，因此输出是int8，在input_workspace中
        ln_compute.ComputeForward(x_data,
                                  ln_scale_data,
                                  ln_bias_data,
                                  input_workspace.data<int8_t>(),
                                  ln_mean_data,
                                  ln_var_data,
                                  nullptr,
                                  0,
                                  qkv_in_scale[i],
                                  quant_round_type,
                                  quant_max_bound,
                                  quant_min_bound);
#ifdef DEBUG_MOE_TMPROFILE_INT8
        dev_ctx.Wait();
        ln_tm.Pause();
#endif
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step2, qkv";
#endif
      // step2. qkv
      const Tensor *qkv_bias = qkv_biases.size() > 0 ? qkv_biases[i] : nullptr;
      // NOTE: in decoder stage, bias is fused in fmha
      const Tensor *bias = time_step ? nullptr : qkv_bias;
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      qkv_tm.Resume();
#endif
      // 输入是int8，input workspace，输出是T，qkv_out
      qkv_compute.ComputeForwardINT8ToT(qkv_weights[i],
                                        qkv_in_scale[i],
                                        &input_workspace,  // input
                                        bias,
                                        &qkv_out,           // out, T
                                        &output_workspace,  // out tmp, int32
                                        &qkv_out,           // bias out, T
                                        qkv_out_scales[i],
                                        &cublaslt_workspace);
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      qkv_tm.Pause();
      fmha_tm.Resume();
#endif
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step3.1 fmha";
#endif
      // step3. fmha
      const Tensor *cache_kv = cache_kvs.size() > 0 ? cache_kvs[i] : nullptr;
      Tensor *cache_kv_out = cache_kv ? cache_kv_outs[i] : nullptr;

      if (time_step) {  // generation decoder stage
        // [2, batch_size, num_head, max_seq_len, head_size]
        int max_seq_len = cache_kv->dims()[3];
        fmha<T>(dev_ctx,
                qkv_out,
                *qkv_bias,
                *src_mask,
                sequence_lengths,
                nullptr,
                beam_cache_offset,
                cache_kv_out,
                &fmha_out,
                bsz,
                beam_size,
                max_seq_len,
                num_head,
                dim_head,
                time_step_cpu,
                0,
                1. / sqrt(dim_head));
      } else if (cache_kv_out) {  // generation context stage
        if (is_support_flash_attn) {
          fmha_fa_compute.ComputeForward(qkv_out,
                                         nullptr,
                                         src_mask,
                                         &transpose_out_2,
                                         nullptr,
                                         &softmax_out,  // softmax_lse_out
                                         &attn_dropout_mask_out,  // seek_offset
                                         &attn_dropout_out,       // softmax_out
                                         &fmha_out);
          // input: [bs, seq_len, 3, num_head, head_dim]
          // output: [3, bs, num_head, seq_len, head_dim]
          std::vector<int> perm_1 = {2, 0, 3, 1, 4};
          transpose_out_2.Resize({{3, bsz, num_head, seq_len, dim_head}});
          TransposeGPUKernelDriver<T>(
              dev_ctx, qkv_out, perm_1, &transpose_out_2);
        } else {
          fmha_compute.ComputeForward(qkv_out,
                                      nullptr,
                                      src_mask,
                                      &transpose_out_2,
                                      nullptr,
                                      &qk_out,
                                      nullptr,
                                      &softmax_out,
                                      &attn_dropout_mask_out,
                                      &attn_dropout_out,
                                      &qktv_out,
                                      &fmha_out);
        }
        // [3, bsz, num_head, seq_len, head_dim]
        T *qkv_data = transpose_out_2_data;
        int64_t q_size = bsz * seq_len * num_head * dim_head;
        int64_t k_size = q_size;
        const T *q_ptr = qkv_data;
        const T *k_ptr = q_ptr + q_size;
        const T *v_ptr = k_ptr + k_size;

        // [2, bsz, num_head, max_seq_len, head_dim]
        int max_seq_len = cache_kv_out->dims()[3];
        T *cache_kv_data = cache_kv_out->data<T>();
        int64_t cache_k_size = bsz * num_head * max_seq_len * dim_head;

        T *cache_k_ptr = cache_kv_data;
        T *cache_v_ptr = cache_kv_data + cache_k_size;

        write_cache_kv<T>(dev_ctx,
                          cache_k_ptr,
                          cache_v_ptr,
                          k_ptr,
                          v_ptr,
                          bsz,
                          num_head,
                          seq_len,
                          max_seq_len,
                          dim_head);
      } else {  // not generation
        VLOG(0) << "not support!";
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step3.2 out linear";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      fmha_tm.Pause();
      out_linear_tm.Resume();
#endif
      // T -> int32
      out_linear_compute.ComputeForwardTToINT8(
          out_linear_weights[i],
          out_linear_in_scale[i],
          &fmha_out,
          &input_workspace,  // input tmp, 先将输入量化
          nullptr,
          &output_workspace,  // output, int32
          nullptr,
          &cublaslt_workspace,
          quant_round_type,
          quant_max_bound,
          quant_min_bound);
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      out_linear_tm.Pause();
#endif
      // 输出在output_workspace
      if (mp_size > 1) {
        phi::AllReduce<int32_t>(output_workspace,
                                ring_id,
                                bsz * seq_len * num_head * dim_head,
                                dev_ctx);
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step4";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      ln_tm.Resume();
#endif
      // step5. ln(residual + dropout(input + bias))
      auto *ln_scale_data = ffn_ln_scales[i]->data<U>();
      auto *ln_bias_data = ffn_ln_biases[i]->data<U>();
      auto *out_linear_bias_data = out_linear_biases[i]->data<T>();
      // input type is int32, src is T, dst is T
      DequantSkipLoadAndStoreResidual<int32_t, T, T, true> load(
          output_workspace.data<int32_t>(),
          out_linear_bias_data,
          x_data,
          out_linear_out_scales[i]->data<float>(),
          bias_dropout_residual_out_data,
          0.0f,
          dim_embed);
      // 改为输出先不做scale，输出是fp16，输出到buf0
      AffineQuantStore<T, LayerNormComputeType, T, false, true> store(
          buf0.data<T>(), dim_embed, ln_scale_data, ln_bias_data);
      DispatchLayerNorm<decltype(load), decltype(store), LayerNormComputeType>(
          dev_ctx.stream(),
          load,
          store,
          bsz_seq,
          dim_embed,
          epsilon,
          ln_mean_data,
          ln_var_data);
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      ln_tm.Pause();
#endif
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step5";
#endif
      // moe
      // step2 resize and slice ln_out
      if (mp_size > 1) {
        sliced_inp = buf0.Slice(start, end);
      } else {
        sliced_inp = buf0;
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, gate & topk";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      gate_tm.Resume();
#endif
      // step3 gate & topk
      // 这里不做量化
      phi::MatMulAndAdd<T>(dev_ctx,
                           gate_weights[i],
                           &sliced_inp,
                           gate_biases[i],
                           false,
                           false,
                           true,  //  compute bias
                           &gate_out,
                           &gate_out);
      phi::TopkKernel<T, phi::GPUContext>(dev_ctx,
                                          gate_out,
                                          topk,  // scalar
                                          -1,
                                          true,
                                          false,
                                          &topk_value,
                                          &topk_idx);
      // step4 prepare forward
      // step4.1 number count
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, number count";
#endif
      phi::NumberCountKernel<int64_t, phi::GPUContext>(
          dev_ctx, topk_idx, tot_expert, &local_expert_count);
      // step4.2 all_to_all
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, all_to_all";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      gate_nccl_tm.Resume();
#endif
      if (world_size > 1) {
        moe_pg.AllToAll<int64_t>(local_expert_count, global_expert_count);
      } else {
        global_expert_count = local_expert_count;
      }
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      gate_nccl_tm.Pause();
#endif
      // global expert count resize
      global_expert_count.Resize({{world_size, num_expert}});
      // fwd expert count
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, fwd expert count";
#endif
      phi::SumKernel<int64_t, phi::GPUContext>(dev_ctx,
                                               global_expert_count,
                                               phi::IntArray({0}),
                                               global_expert_count.dtype(),
                                               false,
                                               &fwd_expert_count);
      // fwd batch size
      phi::SumKernel<int64_t, phi::GPUContext>(
          dev_ctx,
          fwd_expert_count,
          phi::IntArray({}),  // axis is None
          fwd_expert_count.dtype(),
          false,
          &fwd_batch_size);
      // step4.3 cumsum & assign pos
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, cumsum";
#endif
      phi::CumsumKernel<int64_t, phi::GPUContext>(
          dev_ctx, local_expert_count, 0, false, false, false, &lec_cum);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, assign pos";
#endif
      phi::AssignPosCompute<int64_t>(
          dev_ctx, &lec_cum, &topk_idx, &pos, out_batch_size);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, floor divide";
#endif
      if (topk > 1) {
        phi::FloorDivideKernel<int64_t, phi::GPUContext>(
            dev_ctx, pos, topk_tensor, &temp_pos);
      } else {
        temp_pos = pos;
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, tensor copy";
#endif
      framework::TensorCopy(
          fwd_expert_count, platform::CPUPlace(), &fwd_expert_count_cpu);
      framework::TensorCopy(
          fwd_batch_size, platform::CPUPlace(), &fwd_batch_size_cpu);
      dev_ctx.Wait();
      int fwd_bsz = fwd_batch_size_cpu.data<int64_t>()[0];

      Tensor global_scatter_out;
      global_scatter_out.Resize({{fwd_bsz, dim_embed}});
      dev_ctx.Alloc<T>(&global_scatter_out,
                       global_scatter_out.numel() * sizeof(T));

      Tensor all_expert_out;
      all_expert_out.Resize({{fwd_bsz, dim_embed}});
      dev_ctx.Alloc<T>(&all_expert_out, all_expert_out.numel() * sizeof(T));

      // step 5, MOEScatter
      // step 5.1, index select
      // suppose tmp_pos->shape != [0]
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, index select";
#endif
      phi::IndexSelectKernel<T, phi::GPUContext>(
          dev_ctx, sliced_inp, temp_pos, 0, &index_select_out);
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      gate_tm.Pause();

      dev_ctx.Wait();
      scatter_tm.Resume();
#endif
      if (world_size > 1) {
        moe_pg.Scatter<T>(&index_select_out,
                          local_expert_count,
                          global_expert_count,
                          &global_scatter_out);
      } else {
        global_scatter_out = index_select_out;
      }
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      scatter_tm.Pause();

      dev_ctx.Wait();
      expert_tm.Resume();
#endif
      // step 6, Expert Computation
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, Expert Computation";
#endif
      if (fwd_bsz != 0) {
        int last_index = 0;
        for (int idx = 0; idx < num_expert; idx++) {
          int cur_expert_count = fwd_expert_count_cpu.data<int64_t>()[idx];
          if (cur_expert_count <= 0) {
            continue;
          }
          int end = cur_expert_count + last_index;

          Tensor expert_in_tmp;  // int8_t
          expert_in_tmp.Resize({{cur_expert_count, dim_feedforward}});
          dev_ctx.Alloc<int8_t>(&expert_in_tmp,
                                expert_in_tmp.numel() * sizeof(int8_t));

          Tensor expert_out1;  // int32_t
          expert_out1.Resize({{cur_expert_count, dim_feedforward}});
          dev_ctx.Alloc<int32_t>(&expert_out1,
                                 expert_out1.numel() * sizeof(int32_t));

          // input is int32_t, output is int8_t
          FusedDropoutHelper<T, uint8_t, int32_t, int8_t>
              fused_act_dropout_helper(
                  dev_ctx, cur_expert_count, dim_feedforward, dropout_param);

          Tensor tmp_inp =
              global_scatter_out.Slice(last_index, end);  // fp16, T
          int expert_idx = i * num_expert + idx;
          // T to int8_t, matmul, dont compute bias
          MatMulTToINT8<T>(dev_ctx,
                           expert_weights1[expert_idx],
                           expert_weight1_in_scale[expert_idx],
                           &tmp_inp,
                           &expert_in_tmp,
                           &expert_out1,
                           cur_expert_count,
                           dim_feedforward,
                           dim_embed,
                           &cublaslt_workspace,  // maybe space not enough
                           quant_round_type,
                           quant_max_bound,
                           quant_min_bound);
          // act bias, input is int32_t, output is int8_t
          fused_act_dropout_helper.DropoutActBias(
              dev_ctx,
              expert_out1.data<int32_t>(),
              expert_biases1[expert_idx]->data<T>(),
              "gelu",
              expert_in_tmp.data<int8_t>(),  // output
              nullptr,
              expert_weight1_in_scale[expert_idx],
              expert_weight1_out_scales[expert_idx]->data<float>(),
              0,  // data offset
              expert_weight2_in_scale[expert_idx],
              quant_round_type,
              quant_max_bound,
              quant_min_bound,
              approximate);

          // T(fp16)
          Tensor expert_out2 = all_expert_out.Slice(last_index, end);
          // linear2, int8_t to T
          MatMulINT8ToT<T>(dev_ctx,
                           expert_weights2[expert_idx],
                           expert_weight2_in_scale[expert_idx],
                           &expert_in_tmp,  // input
                           expert_biases2[expert_idx],
                           &expert_out2,
                           &expert_out1,  // output_tmp
                           &expert_out2,
                           expert_weight2_out_scales[expert_idx],
                           cur_expert_count,
                           dim_embed,
                           dim_feedforward,
                           true,
                           &cublaslt_workspace);
          last_index = end;
        }
      } else {
        all_expert_out = global_scatter_out;
      }
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      expert_tm.Pause();
      gather_tm.Resume();
#endif
      // step7. MOEGather
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, MOEGather";
#endif
      if (world_size > 1) {
        moe_pg.Gather<T>(&all_expert_out, &global_gather_out);
      } else {
        global_gather_out = all_expert_out;
      }
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      gather_tm.Pause();
#endif
      // step 7.2, local_gather or scatter
      // suppose pos->shape != [0]
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, local_gather or scatter";
#endif
      phi::funcs::GPUScatterAssign<T, int64_t>(
          dev_ctx, global_gather_out, pos, &moe_gather_out, true);

      // step 8, reshape & bmm
      // moe gather out reshape
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, reshape & bmm";
#endif
      moe_gather_out.Resize({{sliced_bsz_seq, topk, dim_embed}});
      topk_value.Resize({{sliced_bsz_seq, 1, topk}});
      phi::BmmKernel<T, phi::GPUContext>(
          dev_ctx, topk_value, moe_gather_out, &bmm_out);
      bmm_out.Resize({{sliced_bsz_seq, dim_embed}});
      // step 9, AllGather
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, AllGather";
#endif
      if (mp_size > 1) {
        // all gather
        moe_pg.AllGather<T>(bmm_out, all_gather_out);
      } else {
        all_gather_out = bmm_out;
      }

      // step 11, add residual
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, add residual";
#endif
      if (i < layers - 1) {
#ifdef DEBUG_MOE_TMPROFILE_INT8
        dev_ctx.Wait();
        ln_tm.Resume();
#endif
        // add residual & next layer norm & qkv quant
        auto *ln_scale_data = ln_scales[i + 1]->data<U>();
        auto *ln_bias_data = ln_biases[i + 1]->data<U>();
        // input type is T, src is T, dst is T
        DequantSkipLoadAndStoreResidual<T, T, T, false> load(
            all_gather_out.data<T>(),
            nullptr,
            bias_dropout_residual_out_data,
            nullptr,
            moe_out.data<T>(),
            0.0f,
            dim_embed);
        AffineQuantStore<int8_t, LayerNormComputeType, T, true, true> store(
            input_workspace.data<int8_t>(),
            dim_embed,
            ln_scale_data,
            ln_bias_data,
            qkv_in_scale[i + 1],
            quant_round_type,
            quant_max_bound,
            quant_min_bound);
        DispatchLayerNorm<decltype(load),
                          decltype(store),
                          LayerNormComputeType>(dev_ctx.stream(),
                                                load,
                                                store,
                                                bsz_seq,
                                                dim_embed,
                                                epsilon,
                                                ln_mean_data,
                                                ln_var_data);
#ifdef DEBUG_MOE_TMPROFILE_INT8
        dev_ctx.Wait();
        ln_tm.Pause();
#endif
      } else {
        // last layer, only add residual, T
        phi::AddKernel<T, phi::GPUContext>(
            dev_ctx, all_gather_out, bias_dropout_residual_out, &moe_out);
      }
      x_data = moe_out.data<T>();
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      trans_tm.Pause();
#endif
    }  // end for layer loop
    out->Resize({{bsz, seq_len, dim_embed}});
#ifdef DEBUG_MOE_TMPROFILE_INT8
    dev_ctx.Wait();
    all_tm.Pause();
    VLOG(0) << "gpu=" << static_cast<int>(dev_ctx.GetPlace().GetDeviceId())
            << ", bsz=" << bsz << ", seq_len=" << seq_len
            << ", total span=" << all_tm.ElapsedMS()
            << ", input=" << other_tm.ElapsedMS()
            << ", transformer=" << trans_tm.ElapsedMS()
            << ", [qkv=" << qkv_tm.ElapsedMS()
            << ", fmha=" << fmha_tm.ElapsedMS()
            << ", out_linear=" << out_linear_tm.ElapsedMS()
            << ", expert=" << expert_tm.ElapsedMS()
            << ", ln=" << ln_tm.ElapsedMS()
            << ", gate/all2all=" << gate_tm.ElapsedMS() << "/"
            << gate_nccl_tm.ElapsedMS()
            << ", scatter=" << scatter_tm.ElapsedMS()
            << ", gather=" << gather_tm.ElapsedMS() << "]";
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    fused_multi_transformer_moe_int8,
    ops::FusedMultiTransformerMoeINT8OpKernel<plat::float16>,
    ops::FusedMultiTransformerMoeINT8OpKernel<float>);