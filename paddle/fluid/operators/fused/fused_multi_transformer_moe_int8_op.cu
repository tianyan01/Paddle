#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
// #define DEBUG_MOE_TMPROFILE_INT8
#include "paddle/fluid/operators/fused/fused_multi_transformer_moe_op.h"
#include "paddle/fluid/operators/fused/fused_multi_transformer_op.h"
#include "paddle/fluid/operators/fused/layernorm_quant_dequant.h"
#include "paddle/phi/kernels/funcs/scatter.cu.h"
#include "paddle/fluid/operators/fused/moe_expert_gemm.h"
#ifdef DEBUG_MOE_TMPROFILE_INT8
#include "paddle/fluid/platform/timer.h"
#endif

DECLARE_bool(enable_moe_gemm_cutlass);

namespace paddle {
namespace operators {
using Tensor = phi::DenseTensor;
// #define _DEBUG_FUSED_MULTI_TRANSFORMER

template <typename T>
static void PrintMatrix(const T *mat_d, int num, std::string name) {
  std::vector<T> tmp(num);
  hipMemcpy(tmp.data(), mat_d, sizeof(T) * num, hipMemcpyDeviceToHost);

  std::ofstream outfile;
  outfile.open(name + ".txt", std::ios::out);
  std::stringstream ss;

  for (int i = 0; i < num; ++i) {
    if (std::is_same<T, int8_t>::value) {
      ss << static_cast<int>(tmp[i]) << std::endl;
    } else {
      ss << std::setprecision(8) << tmp[i] << std::endl;
    }
  }
  outfile << ss.str();
  outfile.close();
}

inline bool CheckFlashAttn(const phi::GPUContext &dev_ctx,
                           const phi::DenseTensor &x) {
  int dev = dev_ctx.GetPlace().GetDeviceId();
  if (!paddle::platform::IsSupportFlashAttn(dev)) {
    return false;
  }
  return (x.dtype() == DataType::FLOAT16);
}
template <typename T>
class FusedMultiTransformerMoeINT8OpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    auto &dev_ctx = ctx.cuda_device_context();
#ifdef DEBUG_MOE_TMPROFILE_INT8
    platform::Timer all_tm, other_tm, trans_tm;
    platform::Timer qkv_tm, fmha_tm, out_linear_tm;
    platform::Timer expert_tm, ln_tm, gate_tm;
    platform::Timer gate_nccl_tm, gather_tm, scatter_tm;
    all_tm.Start();
    other_tm.Start();
#endif
#ifndef PADDLE_WITH_CUTLASS
    PADDLE_ENFORCE_EQ(FLAGS_enable_moe_gemm_cutlass, false,
                      "not support cutlass fused moe gemm please disable "
                      "FLAGS_enable_moe_gemm_cutlass");
#endif
    auto *time_step = ctx.Input<Tensor>("TimeStep");
    // 0. input
    auto *input_x = ctx.Input<Tensor>("X");
    const auto input_x_dims = input_x->dims();
    int bsz = input_x_dims[0];
    int seq_len = input_x_dims[1];
    int dim_embed = input_x_dims[2];
    int bsz_seq = bsz * seq_len;
    if (bsz_seq == 0) {
      return;
    }

    // quant input scales, vector, size = num_layers
    auto qkv_in_scale = ctx.Attr<std::vector<float>>("qkv_in_scale");
    auto out_linear_in_scale =
        ctx.Attr<std::vector<float>>("out_linear_in_scale");
    // moe expert scales, vector, size = num_expert * num_layers
    auto expert_weight1_in_scale =
        ctx.Attr<std::vector<float>>("expert_weight1_in_scale");
    auto expert_weight2_in_scale =
        ctx.Attr<std::vector<float>>("expert_weight2_in_scale");

    // quant round type and bound
    auto quant_round_type = ctx.Attr<int>("quant_round_type");
    auto quant_max_bound = ctx.Attr<float>("quant_max_bound");
    auto quant_min_bound = ctx.Attr<float>("quant_min_bound");

    // dequant output scales, vertor<tensor>, size = [num_layers, n], n is gemm
    // output size
    auto qkv_out_scales = ctx.MultiInput<Tensor>("QKVOutScale");
    auto out_linear_out_scales = ctx.MultiInput<Tensor>("OutLinearOutScale");
    // dequant output scales, tensor, size = [num_layers * num_expert, n], n is
    // gemm output size
    auto expert_weight1_out_scales =
        ctx.MultiInput<Tensor>("ExpertWeight1OutScale");
    auto expert_weight2_out_scales =
        ctx.MultiInput<Tensor>("ExpertWeight2OutScale");

    bool encoder_remove_padding = false;
    auto *sequence_lengths = ctx.Input<Tensor>("SeqLengths");
    if (sequence_lengths && !time_step) {
      encoder_remove_padding = true;
    }
    Tensor d_token_tensor;
    Tensor padding_offset_tensor;
    Tensor x_remove_padding;
    int token_num = 0;

    auto *beam_cache_offset = ctx.Input<Tensor>("BeamCacheOffset");
    int beam_size = 1;
    if (beam_cache_offset) {
      beam_size = beam_cache_offset->dims()[1];
    }

    auto *out = ctx.Output<Tensor>("Out");
    auto *from_data = dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));
    // Init out & remove padding in encoder
    if (encoder_remove_padding) {
      InitValue(dev_ctx, from_data, out->numel(), static_cast<T>(0.));
      // just for encoder
      d_token_tensor.Resize({{1}});
      auto *d_token_num = dev_ctx.Alloc<int>(
          &d_token_tensor, d_token_tensor.numel() * sizeof(int));
      // alloc the max size of padding_offset_tensor
      padding_offset_tensor.Resize({{bsz_seq}});
      dev_ctx.Alloc<int>(&padding_offset_tensor,
                         padding_offset_tensor.numel() * sizeof(int));
      InvokeGetPaddingOffset(dev_ctx,
                             &token_num,
                             d_token_num,
                             padding_offset_tensor.data<int>(),
                             sequence_lengths->data<int>(),
                             bsz,
                             seq_len);
      padding_offset_tensor.Resize({{token_num}});
      x_remove_padding.Resize({{token_num, dim_embed}});
      dev_ctx.Alloc<T>(&x_remove_padding, x_remove_padding.numel() * sizeof(T));
      InvokeRemovePadding(dev_ctx,
                          x_remove_padding.data<T>(),
                          input_x->data<T>(),
                          padding_offset_tensor.data<int>(),
                          token_num,
                          dim_embed);
    } else {
      token_num = bsz_seq;
    }
    auto *padding_offset_data =
        encoder_remove_padding ? padding_offset_tensor.data<int>() : nullptr;

    // 1. layer norm
    const auto pre_layer_norm = ctx.Attr<bool>("pre_layer_norm");
    if (!pre_layer_norm) {
      VLOG(0) << "not support post layer norm!";
      return;
    }
    const float epsilon = ctx.Attr<float>("epsilon");
    auto ln_scales = ctx.MultiInput<Tensor>("LnScale");
    auto ln_biases = ctx.MultiInput<Tensor>("LnBias");

    // in type is T, out type is int8_t
    auto ln_compute =
        AttnLayerNorm<T, T, int8_t>(dev_ctx, epsilon, token_num, dim_embed);
    Tensor ln_mean, ln_var;
    ln_mean.Resize({{token_num}});
    auto *ln_mean_data =
        dev_ctx.Alloc<U>(&ln_mean, ln_mean.numel() * sizeof(U));
    ln_var.Resize({{token_num}});
    auto *ln_var_data = dev_ctx.Alloc<U>(&ln_var, ln_var.numel() * sizeof(U));

    // 2. qkv
    // x: qkv's input [batch_size, seq_len, dim_embed]
    // y: qkv's weight: [3, num_head, dim_head, dim_embed]
    auto qkv_weights = ctx.MultiInput<Tensor>("QKVW");
    auto qkv_biases = ctx.MultiInput<Tensor>("QKVBias");
    const bool trans_qkvw = ctx.Attr<bool>("trans_qkvw");
    const auto qkv_w_dims = qkv_weights[0]->dims();
    int num_head = trans_qkvw ? qkv_w_dims[1] : qkv_w_dims[2];
    int dim_head = trans_qkvw ? qkv_w_dims[2] : qkv_w_dims[3];
    int hidden_size = num_head * dim_head;
    int output_size = 3 * hidden_size;
    int input_size = dim_embed;

    bool compute_bias = qkv_biases.size() > 0 && time_step == nullptr;
    // (transA, transB, compute_bias) = (false, trans_qkvw, false)
    AttnMatmulINT8<T> qkv_compute(
        dev_ctx, token_num, output_size, input_size, false);
    Tensor qkv_out;
    qkv_out.Resize({{bsz_seq, 3, num_head, dim_head}}); // token_num
    auto *qkv_out_data =
        dev_ctx.Alloc<T>(&qkv_out, qkv_out.numel() * sizeof(T));

    // 3. fmha
    // check support flash attn
    AttnDropoutParam attn_param(
        true, "upscale_in_train", 0.0, true, true, 0, nullptr);
    bool is_support_flash_attn = CheckFlashAttn(dev_ctx, *input_x);
    auto fmha_fa_compute = FlashAttnFMHARef<plat::float16>(
        dev_ctx, bsz, seq_len, num_head, dim_head, attn_param);
    auto fmha_compute =
        FMHARef<T>(dev_ctx, bsz, seq_len, num_head, dim_head, attn_param);
    auto *src_mask = ctx.Input<Tensor>("SrcMask");
    auto cache_kvs = ctx.MultiInput<Tensor>("CacheKV");
    auto cache_kv_outs = ctx.MultiOutput<Tensor>("CacheKVOut");

    int time_step_cpu = 0;
    auto out_seq_len = seq_len;
    if (time_step) {
      time_step_cpu = src_mask->dims()[3] - 1;
      out_seq_len += time_step_cpu;
    }

    Tensor transpose_out_2, qk_out;
    transpose_out_2.Resize({{3, bsz, num_head, seq_len, dim_head}});
    auto *transpose_out_2_data =
        dev_ctx.Alloc<T>(&transpose_out_2, transpose_out_2.numel() * sizeof(T));
    auto *q_transpose_out_data = transpose_out_2_data;
    auto *kv_transpose_out_data = transpose_out_2_data + bsz * seq_len * dim_embed;

    Tensor q_transpose_out = transpose_out_2.Slice(0, 1);

    if (encoder_remove_padding) {
      InitValue(dev_ctx,
                transpose_out_2_data,
                transpose_out_2.numel(),
                static_cast<T>(0.));
    }

    Tensor softmax_out;
    Tensor attn_dropout_mask_out, attn_dropout_out;
    Tensor qktv_out, fmha_out;
    if (!is_support_flash_attn) {
      qk_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
      auto *qk_out_data = dev_ctx.Alloc<T>(&qk_out, qk_out.numel() * sizeof(T));

      softmax_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
      auto *softmax_out_data =
          dev_ctx.Alloc<T>(&softmax_out, softmax_out.numel() * sizeof(T));

      qktv_out.Resize({{bsz, num_head, seq_len, dim_head}});
      auto *qktv_out_data =
          dev_ctx.Alloc<T>(&qktv_out, qktv_out.numel() * sizeof(T));
    }

    fmha_out.Resize({{bsz, seq_len, num_head, dim_head}});
    auto *fmha_out_data =
        dev_ctx.Alloc<T>(&fmha_out, fmha_out.numel() * sizeof(T));

    // 4. out_linear  注意！！这个weight的维度需要transpose！！
    auto out_linear_weights = ctx.MultiInput<Tensor>("OutLinearW");
    auto out_linear_biases = ctx.MultiInput<Tensor>("OutLinearBias");
    int ring_id = ctx.Attr<int>("ring_id");
    // (transA, transB, compute_bias) = (false, false, false)
    AttnMatmulINT8<T> out_linear_compute(
        dev_ctx, token_num, dim_embed, hidden_size, false);

    // 5. ln(residual + bias)
    DropoutParam dropout_param(false, 0, true, true, 0.0, nullptr, 0);

    using LayerNormComputeType = float;
    auto ffn_ln_scales = ctx.MultiInput<Tensor>("FFNLnScale");
    auto ffn_ln_biases = ctx.MultiInput<Tensor>("FFNLnBias");
    Tensor bias_dropout_residual_out, dropout_mask_out;
    T *bias_dropout_residual_out_data = nullptr;
    bias_dropout_residual_out.Resize({{token_num, dim_embed}});
    bias_dropout_residual_out_data =
        dev_ctx.Alloc<T>(&bias_dropout_residual_out,
                         bias_dropout_residual_out.numel() * sizeof(T));

    // 6. moe layer: gate / expert_w & b / some attrs
    auto gate_weights = ctx.MultiInput<Tensor>("GateWeight");
    auto gate_biases = ctx.MultiInput<Tensor>("GateBias");
    // weight的维度需要transpose！！！！
    auto expert_weights1 = ctx.MultiInput<Tensor>("ExpertWeight1");
    auto expert_biases1 = ctx.MultiInput<Tensor>("ExpertBias1");
    auto expert_weights2 = ctx.MultiInput<Tensor>("ExpertWeight2");
    auto expert_biases2 = ctx.MultiInput<Tensor>("ExpertBias2");
    int dim_feedforward =
        expert_weights1[0]->dims()[0];  // dim is [dim_feedforward, dim_embed]
    int topk = ctx.Attr<int>("topk");
    int mp_size = ctx.Attr<int>("mp_size");
    int mp_rank = ctx.Attr<int>("mp_rank");
    int num_expert = ctx.Attr<int>("num_expert");
    int world_size = ctx.Attr<int>("world_size");
    int moe_ring_id = ctx.Attr<int>("moe_ring_id");
    bool approximate = ctx.Attr<bool>("approximate");
    if (encoder_remove_padding) {
      PADDLE_ENFORCE_EQ(mp_size, 1,
                      "When encoder remove padding, mp_size should be 1!");
    }

    int tot_expert = world_size * num_expert;
    // after slice, bsz_seq should be change
    int sliced_bsz_seq = token_num;
    int start = 0;
    int end = 0;
    if (mp_size > 1) {
      start = token_num / world_size * mp_rank;
      end = std::min(start + token_num / world_size, token_num);
      sliced_bsz_seq = end - start;
    }
    int out_batch_size = sliced_bsz_seq * topk;
    // slice
    Tensor sliced_inp;
    sliced_inp.Resize({{sliced_bsz_seq, dim_embed}});
    dev_ctx.Alloc<T>(&sliced_inp, sliced_inp.numel() * sizeof(T));
    // gate linear
    Tensor gate_out;
    gate_out.Resize({{sliced_bsz_seq, tot_expert}});
    dev_ctx.Alloc<T>(&gate_out, gate_out.numel() * sizeof(T));
    // topk
    Tensor topk_value, topk_idx;
    topk_value.Resize({{sliced_bsz_seq, topk}});
    dev_ctx.Alloc<T>(&topk_value, topk_value.numel() * sizeof(T));
    topk_idx.Resize({{sliced_bsz_seq, topk}});
    dev_ctx.Alloc<int64_t>(&topk_idx, topk_idx.numel() * sizeof(int64_t));
    // local expert count, global expert count
    Tensor local_expert_count, global_expert_count;
    local_expert_count.Resize({{tot_expert}});
    global_expert_count.Resize({{tot_expert}});
    dev_ctx.Alloc<int64_t>(&local_expert_count,
                           local_expert_count.numel() * sizeof(int64_t));
    dev_ctx.Alloc<int64_t>(&global_expert_count,
                           global_expert_count.numel() * sizeof(int64_t));
    // fwd_expert_count
    Tensor fwd_expert_count, fwd_expert_count_cumsum;
    Tensor fwd_expert_count_cumsum_cpu;
    fwd_expert_count.Resize({{num_expert}});
    fwd_expert_count_cumsum.Resize({{num_expert + 1}});
    dev_ctx.Alloc<int64_t>(&fwd_expert_count,
                           fwd_expert_count.numel() * sizeof(int64_t));
    auto fwd_expert_count_cumsum_data = dev_ctx.Alloc<int64_t>(&fwd_expert_count_cumsum,
                                            fwd_expert_count_cumsum.numel() * sizeof(int64_t));
    phi::funcs::set_constant<int64_t>(
        dev_ctx, &fwd_expert_count_cumsum, static_cast<int64_t>(0));
    // pos, temp pos
    Tensor pos, temp_pos;
    pos.Resize({{out_batch_size}});
    temp_pos.Resize({{out_batch_size}});
    dev_ctx.Alloc<int64_t>(&pos, pos.numel() * sizeof(int64_t));
    if (topk > 1) {
      dev_ctx.Alloc<int64_t>(&temp_pos, temp_pos.numel() * sizeof(int64_t));
    }
    // cumsum
    Tensor lec_cum;
    lec_cum.Resize({{tot_expert}});
    dev_ctx.Alloc<int64_t>(&lec_cum, lec_cum.numel() * sizeof(int64_t));
    // fused moe ffn tmp out
    Tensor index_select_out;
    index_select_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&index_select_out, index_select_out.numel() * sizeof(T));
    Tensor global_gather_out;
    global_gather_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&global_gather_out, global_gather_out.numel() * sizeof(T));
    Tensor moe_gather_out;
    moe_gather_out.Resize({{out_batch_size, dim_embed}});
    dev_ctx.Alloc<T>(&moe_gather_out, moe_gather_out.numel() * sizeof(T));
    Tensor bmm_out;
    bmm_out.Resize({{sliced_bsz_seq, 1, dim_embed}});
    dev_ctx.Alloc<T>(&bmm_out, bmm_out.numel() * sizeof(T));
    Tensor all_gather_out;
    all_gather_out.Resize({{token_num, dim_embed}});
    dev_ctx.Alloc<T>(&all_gather_out, all_gather_out.numel() * sizeof(T));

    // moe nccl
    phi::NCCLMoECollective moe_pg(dev_ctx, moe_ring_id, num_expert);

    // []. init workspace for cublasLt transform
    Tensor input_workspace, output_workspace, cublaslt_workspace;
    // for input and output transform data is CUBLASLT_ORDER_COL32 format,
    int m_max = token_num, k_max = std::max({dim_embed, dim_feedforward}),
        n_max = std::max({output_size, dim_embed, dim_feedforward});
    // maybe need to change the size of workspace here

    input_workspace.Resize({{(m_max * k_max + 31) / 32 * 32}});
    dev_ctx.Alloc<int8_t>(&input_workspace,
                          input_workspace.numel() * sizeof(int8_t));

    output_workspace.Resize({{(n_max * m_max + 31) / 32 * 32}});
    dev_ctx.Alloc<int32_t>(&output_workspace,
                           output_workspace.numel() * sizeof(int32_t));

    cublaslt_workspace.Resize({{3000000}});
    dev_ctx.Alloc<int8_t>(&cublaslt_workspace,
                          cublaslt_workspace.numel() * sizeof(int8_t));

    // calc
    Tensor buf0;
    buf0.Resize({{token_num, dim_embed}});
    dev_ctx.Alloc<T>(&buf0, buf0.numel() * sizeof(T));

    Tensor *moe_out = nullptr;
    if (encoder_remove_padding) {
      moe_out = &buf0;
    } else {
      moe_out = out;
      moe_out->Resize({{token_num, dim_embed}});
    }

    const T *x_data = encoder_remove_padding ? x_remove_padding.data<T>() : input_x->data<T>();

#ifdef DEBUG_MOE_TMPROFILE_INT8
    dev_ctx.Wait();
    other_tm.Pause();
#endif
    int layers = qkv_weights.size();
    for (int i = 0; i < layers; ++i) {
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step1, pre layernorm";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      trans_tm.Resume();
#endif
      // step1. layer_norm
      if (i == 0) {
#ifdef DEBUG_MOE_TMPROFILE_INT8
        ln_tm.Resume();
#endif
        auto *ln_scale_data = ln_scales[i]->data<U>();
        auto *ln_bias_data = ln_biases[i]->data<U>();
        // layer norm后，对输出做scale，因此输出是int8，在input_workspace中
        ln_compute.ComputeForward(x_data,
                                  ln_scale_data,
                                  ln_bias_data,
                                  input_workspace.data<int8_t>(),
                                  ln_mean_data,
                                  ln_var_data,
                                  nullptr,
                                  0,
                                  qkv_in_scale[i],
                                  quant_round_type,
                                  quant_max_bound,
                                  quant_min_bound);
#ifdef DEBUG_MOE_TMPROFILE_INT8
        dev_ctx.Wait();
        ln_tm.Pause();
#endif
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step2, qkv";
#endif
      // step2. qkv
      const Tensor *qkv_bias = qkv_biases.size() > 0 ? qkv_biases[i] : nullptr;
      // NOTE: in decoder stage, bias is fused in fmha
      const Tensor *bias = time_step ? nullptr : qkv_bias;
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      qkv_tm.Resume();
#endif
      // 输入是int8，input workspace，输出是T，qkv_out
      qkv_compute.ComputeForwardINT8ToT(qkv_weights[i],
                                        qkv_in_scale[i],
                                        &input_workspace,  // input
                                        bias,
                                        &qkv_out,           // out, T
                                        &output_workspace,  // out tmp, int32
                                        &qkv_out,           // bias out, T
                                        qkv_out_scales[i],
                                        &cublaslt_workspace);
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      qkv_tm.Pause();
      fmha_tm.Resume();
#endif
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step3.1 fmha";
#endif
      // step3. fmha
      const Tensor *cache_kv = cache_kvs.size() > 0 ? cache_kvs[i] : nullptr;
      Tensor *cache_kv_out = cache_kv ? cache_kv_outs[i] : nullptr;
      // [2, batch_size, num_head, max_seq_len, head_size]
      int max_seq_len = cache_kv->dims()[3];

      if (time_step) {  // generation decoder stage
        fmha<T>(dev_ctx,
                qkv_out,
                *qkv_bias,
                *src_mask,
                sequence_lengths,
                nullptr,
                beam_cache_offset,
                cache_kv_out,
                &fmha_out,
                bsz,
                beam_size,
                max_seq_len,
                num_head,
                dim_head,
                time_step_cpu,
                0,
                1. / sqrt(dim_head));
      } else if (cache_kv_out) {  // generation context stage, encoder
        // transpose(qkv_out + qkv_bias)
        qkv_bias_add_transpose_split<T>(dev_ctx,
                                        q_transpose_out_data,
                                        kv_transpose_out_data,
                                        qkv_out_data,
                                        qkv_bias->data<T>(),
                                        padding_offset_data,
                                        token_num,
                                        bsz,
                                        num_head,
                                        seq_len,
                                        dim_head,
                                        compute_bias);
        // first write cache kv
        // const T *q_ptr = q_transpose_out_data;
        const T *k_ptr = kv_transpose_out_data;
        const T *v_ptr = k_ptr + bsz * seq_len * num_head * dim_head;;

        T *cache_kv_data = cache_kv_out->data<T>();
        int64_t cache_k_size = bsz * num_head * max_seq_len * dim_head;

        T *cache_k_ptr = cache_kv_data;
        T *cache_v_ptr = cache_kv_data + cache_k_size;

        const int *sequence_lengths_data =
            encoder_remove_padding ? sequence_lengths->data<int>() : nullptr;
        write_cache_kv<T>(dev_ctx,
                          cache_k_ptr,
                          cache_v_ptr,
                          k_ptr,
                          v_ptr,
                          sequence_lengths_data,
                          bsz,
                          num_head,
                          seq_len,
                          max_seq_len,
                          dim_head);
        phi::DenseTensor *tmp_padding_offset_tensor =
            encoder_remove_padding ? &padding_offset_tensor : nullptr;
        // compute q * kt * v
        if (is_support_flash_attn) {
          qkv_out.Resize({{3, bsz, seq_len, num_head, dim_head}});
          fmha_fa_compute.RemovePaddingComputeForward(src_mask,
                                                      tmp_padding_offset_tensor,
                                                      &transpose_out_2,
                                                      &qkv_out,       // real input
                                                      &softmax_out,  // softmax_lse_out
                                                      &attn_dropout_mask_out,  // seek_offset
                                                      &attn_dropout_out,       // softmax_out
                                                      &q_transpose_out,        // tmp buf
                                                      &fmha_out,     // output
                                                      token_num);
        } else {
          // Tensor q_transpose_out = transpose_out_2.Slice(0, 1);
          Tensor kv_transpose_out = transpose_out_2.Slice(1, 3);
          fmha_compute.ComputeForwardWithoutTranspose(nullptr,
                                                      src_mask,
                                                      tmp_padding_offset_tensor,
                                                      &q_transpose_out,
                                                      &kv_transpose_out,
                                                      nullptr,
                                                      &qk_out,
                                                      nullptr,
                                                      &softmax_out,
                                                      &attn_dropout_mask_out,
                                                      &attn_dropout_out,
                                                      &qktv_out,
                                                      &fmha_out,
                                                      token_num);
        }
      } else {  // not generation
        VLOG(0) << "not support!";
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step3.2 out linear";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      fmha_tm.Pause();
      out_linear_tm.Resume();
#endif
      // T -> int32
      out_linear_compute.ComputeForwardTToINT8(
          out_linear_weights[i],
          out_linear_in_scale[i],
          &fmha_out,
          &input_workspace,  // input tmp, 先将输入量化
          nullptr,
          &output_workspace,  // output, int32
          nullptr,
          &cublaslt_workspace,
          quant_round_type,
          quant_max_bound,
          quant_min_bound);
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      out_linear_tm.Pause();
#endif
      // 输出在output_workspace
      if (mp_size > 1) {
        phi::AllReduce<int32_t>(output_workspace,
                                ring_id,
                                bsz * seq_len * num_head * dim_head,
                                dev_ctx);
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step4";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      ln_tm.Resume();
#endif
      // step5. ln(residual + dropout(input + bias))
      auto *ln_scale_data = ffn_ln_scales[i]->data<U>();
      auto *ln_bias_data = ffn_ln_biases[i]->data<U>();
      auto *out_linear_bias_data = out_linear_biases[i]->data<T>();
      // input type is int32, src is T, dst is T
      DequantSkipLoadAndStoreResidual<int32_t, T, T, true> load(
          output_workspace.data<int32_t>(),
          out_linear_bias_data,
          x_data,
          out_linear_out_scales[i]->data<float>(),
          bias_dropout_residual_out_data,
          0.0f,
          dim_embed);
      // 改为输出先不做scale，输出是fp16，输出到buf0 [token_num, dim_emb]
      AffineQuantStore<T, LayerNormComputeType, T, false, true> store(
          buf0.data<T>(), dim_embed, ln_scale_data, ln_bias_data);
      DispatchLayerNorm<decltype(load), decltype(store), LayerNormComputeType>(
          dev_ctx.stream(),
          load,
          store,
          token_num,
          dim_embed,
          epsilon,
          ln_mean_data,
          ln_var_data);
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      ln_tm.Pause();
#endif
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "step5";
#endif
      // moe
      // step2 resize and slice ln_out
      if (mp_size > 1) {
        sliced_inp = buf0.Slice(start, end);
      } else {
        sliced_inp = buf0;
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, gate & topk";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      gate_tm.Resume();
#endif
      // step3 gate & topk
      // 这里不做量化
      phi::MatMulAndAdd<T>(dev_ctx,
                           gate_weights[i],
                           &sliced_inp,
                           gate_biases[i],
                           false,
                           false,
                           true,  //  compute bias
                           &gate_out,
                           &gate_out);
      phi::TopkKernel<T, phi::GPUContext>(dev_ctx,
                                          gate_out,
                                          topk,  // scalar
                                          -1,
                                          true,
                                          false,
                                          &topk_value,
                                          &topk_idx);
      // step4 prepare forward
      // step4.1 number count
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, number count";
#endif
      phi::NumberCountKernel<int64_t, phi::GPUContext>(
          dev_ctx, topk_idx, tot_expert, &local_expert_count);
      // step4.2 all_to_all
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, all_to_all";
#endif
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      gate_nccl_tm.Resume();
#endif
      if (world_size > 1) {
        moe_pg.AllToAll<int64_t>(local_expert_count, global_expert_count);
      } else {
        global_expert_count = local_expert_count;
      }
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      gate_nccl_tm.Pause();
#endif
      // global expert count resize
      global_expert_count.Resize({{world_size, num_expert}});
      // fwd expert count
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, fwd expert count";
#endif
      phi::SumKernel<int64_t, phi::GPUContext>(dev_ctx,
                                               global_expert_count,
                                               phi::IntArray({0}),
                                               global_expert_count.dtype(),
                                               false,
                                               &fwd_expert_count);
      // fwd batch size, we dont compute this
      phi::CumsumTensorValue<int64_t>(
          dev_ctx, fwd_expert_count, &fwd_expert_count_cumsum, 1);
      // step4.3 cumsum & assign pos
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, cumsum";
#endif
      phi::CumsumTensorValue<int64_t>(dev_ctx, local_expert_count, &lec_cum);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, assign pos";
#endif
      phi::AssignInsAndPosCompute<int64_t>(
          dev_ctx, &lec_cum, &topk_idx, &pos, out_batch_size, topk, &temp_pos);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, tensor copy";
#endif
      framework::TensorCopy(
          fwd_expert_count_cumsum, platform::CPUPlace(), &fwd_expert_count_cumsum_cpu);
      dev_ctx.Wait();
      int fwd_bsz = fwd_expert_count_cumsum_cpu.data<int64_t>()[num_expert];

      Tensor global_scatter_out;
      global_scatter_out.Resize({{fwd_bsz, dim_embed}});
      auto global_scatter_out_data = dev_ctx.Alloc<T>(&global_scatter_out,
                                        global_scatter_out.numel() * sizeof(T));

      Tensor all_expert_out;
      all_expert_out.Resize({{fwd_bsz, dim_embed}});
      auto all_expert_out_data = dev_ctx.Alloc<T>(&all_expert_out, 
                                    all_expert_out.numel() * sizeof(T));

      // step 5, MOEScatter
      // step 5.1, index select
      // suppose tmp_pos->shape != [0]
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, index select";
#endif
      phi::IndexSelectKernel<T, phi::GPUContext>(
          dev_ctx, sliced_inp, temp_pos, 0, &index_select_out);
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      gate_tm.Pause();

      dev_ctx.Wait();
      scatter_tm.Resume();
#endif
      if (world_size > 1) {
        moe_pg.Scatter<T>(&index_select_out,
                          local_expert_count,
                          global_expert_count,
                          &global_scatter_out);
      } else {
        global_scatter_out = index_select_out;
      }
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      scatter_tm.Pause();

      dev_ctx.Wait();
      expert_tm.Resume();
#endif
      // step 6, Expert Computation
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, Expert Computation";
#endif
      if (fwd_bsz != 0) {
        if (FLAGS_enable_moe_gemm_cutlass) {
          // grouped gemm
          int expert_idx = i * num_expert;
          // expert
          Tensor expert_in_tmp, expert_out1;  // int8_t, int32_t
          expert_in_tmp.Resize({{fwd_bsz, dim_feedforward}});
          auto expert_in_tmp_data = dev_ctx.Alloc<int8_t>(&expert_in_tmp,
                                        expert_in_tmp.numel() * sizeof(int8_t));

          expert_out1.Resize({{fwd_bsz, dim_feedforward}});
          auto expert_out1_data = dev_ctx.Alloc<T>(&expert_out1,
                                      expert_out1.numel() * sizeof(T)); // dequant 输出, fp16
          // gemm1, do act
          FusedGroupedMatMul(dev_ctx,
                             expert_weights1[expert_idx]->data<int8_t>(),
                             global_scatter_out_data,
                             &expert_in_tmp,
                             &expert_weight1_in_scale[expert_idx],
                             expert_biases1[expert_idx]->data<T>(),
                             expert_out1_data, // dequant & bias & gelu output
                             expert_weight1_out_scales[expert_idx]->data<float>(),
                             fwd_expert_count_cumsum_data,
                             fwd_expert_count_cumsum_cpu.data<int64_t>(),
                             fwd_bsz,
                             num_expert,
                             fwd_bsz,
                             dim_feedforward,
                             dim_embed,
                             true);
          // gemm2, no act
          FusedGroupedMatMul(dev_ctx,
                             expert_weights2[expert_idx]->data<int8_t>(),
                             expert_out1_data,
                             &expert_in_tmp,
                             &expert_weight2_in_scale[expert_idx],
                             expert_biases2[expert_idx]->data<T>(),
                             all_expert_out_data, // dequant output
                             expert_weight2_out_scales[expert_idx]->data<float>(),
                             fwd_expert_count_cumsum_data,
                             fwd_expert_count_cumsum_cpu.data<int64_t>(),
                             fwd_bsz,
                             num_expert,
                             fwd_bsz,
                             dim_embed,
                             dim_feedforward,
                             false);
        } else {
          int last_index = 0;
          int64_t *csum_len = fwd_expert_count_cumsum_cpu.data<int64_t>();
          for (int idx = 0; idx < num_expert; idx++) {
            int end = csum_len[idx + 1];
            int cur_expert_count = end - last_index;
            if (cur_expert_count <= 0) {
              continue;
            }

            Tensor expert_in_tmp;  // int8_t
            expert_in_tmp.Resize({{cur_expert_count, dim_feedforward}});
            dev_ctx.Alloc<int8_t>(&expert_in_tmp,
                                  expert_in_tmp.numel() * sizeof(int8_t));

            Tensor expert_out1;  // int32_t
            expert_out1.Resize({{cur_expert_count, dim_feedforward}});
            dev_ctx.Alloc<int32_t>(&expert_out1,
                                  expert_out1.numel() * sizeof(int32_t));

            // input is int32_t, output is int8_t
            FusedDropoutHelper<T, uint8_t, int32_t, int8_t>
                fused_act_dropout_helper(
                    dev_ctx, cur_expert_count, dim_feedforward, dropout_param);

            Tensor tmp_inp =
                global_scatter_out.Slice(last_index, end);  // fp16, T
            int expert_idx = i * num_expert + idx;
            // T to int8_t, matmul, dont compute bias
            MatMulTToINT8<T>(dev_ctx,
                            expert_weights1[expert_idx],
                            expert_weight1_in_scale[expert_idx],
                            &tmp_inp,
                            &expert_in_tmp,
                            &expert_out1,
                            cur_expert_count,
                            dim_feedforward,
                            dim_embed,
                            &cublaslt_workspace,  // maybe space not enough
                            quant_round_type,
                            quant_max_bound,
                            quant_min_bound);
            // act bias, input is int32_t, output is int8_t
            fused_act_dropout_helper.DropoutActBias(
                dev_ctx,
                expert_out1.data<int32_t>(),
                expert_biases1[expert_idx]->data<T>(),
                "gelu",
                expert_in_tmp.data<int8_t>(),  // output
                nullptr,
                expert_weight1_in_scale[expert_idx],
                expert_weight1_out_scales[expert_idx]->data<float>(),
                0,  // data offset
                expert_weight2_in_scale[expert_idx],
                quant_round_type,
                quant_max_bound,
                quant_min_bound,
                approximate);

            // T(fp16)
            Tensor expert_out2 = all_expert_out.Slice(last_index, end);
            // linear2, int8_t to T
            MatMulINT8ToT<T>(dev_ctx,
                            expert_weights2[expert_idx],
                            expert_weight2_in_scale[expert_idx],
                            &expert_in_tmp,  // input
                            expert_biases2[expert_idx],
                            &expert_out2,
                            &expert_out1,  // output_tmp
                            &expert_out2,
                            expert_weight2_out_scales[expert_idx],
                            cur_expert_count,
                            dim_embed,
                            dim_feedforward,
                            true,
                            &cublaslt_workspace);
            last_index = end;
          }
        }
      } else {
        all_expert_out = global_scatter_out;
      }
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      expert_tm.Pause();
      gather_tm.Resume();
#endif
      // step7. MOEGather
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, MOEGather";
#endif
      if (world_size > 1) {
        moe_pg.Gather<T>(&all_expert_out, &global_gather_out);
      } else {
        global_gather_out = all_expert_out;
      }
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      gather_tm.Pause();
#endif
      // step 7.2, local_gather or scatter
      // suppose pos->shape != [0]
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, local_gather or scatter";
#endif
      phi::funcs::GPUScatterAssign<T, int64_t>(
          dev_ctx, global_gather_out, pos, &moe_gather_out, true);

      // step 8, reshape & bmm
      // moe gather out reshape
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, reshape & bmm";
#endif
      moe_gather_out.Resize({{sliced_bsz_seq, topk, dim_embed}});
      topk_value.Resize({{sliced_bsz_seq, 1, topk}});
      phi::BmmKernel<T, phi::GPUContext>(
          dev_ctx, topk_value, moe_gather_out, &bmm_out);
      bmm_out.Resize({{sliced_bsz_seq, dim_embed}});
      // step 9, AllGather
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, AllGather";
#endif
      if (mp_size > 1) {
        // all gather
        moe_pg.AllGather<T>(bmm_out, all_gather_out);
      } else {
        all_gather_out = bmm_out;
      }

      // step 11, add residual
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(0) << "moe, add residual";
#endif
      if (i < layers - 1) {
#ifdef DEBUG_MOE_TMPROFILE_INT8
        dev_ctx.Wait();
        ln_tm.Resume();
#endif
        // add residual & next layer norm & qkv quant
        auto *ln_scale_data = ln_scales[i + 1]->data<U>();
        auto *ln_bias_data = ln_biases[i + 1]->data<U>();
        // input type is T, src is T, dst is T
        DequantSkipLoadAndStoreResidual<T, T, T, false> load(
            all_gather_out.data<T>(),
            nullptr,
            bias_dropout_residual_out_data,
            nullptr,
            moe_out->data<T>(),
            0.0f,
            dim_embed);
        AffineQuantStore<int8_t, LayerNormComputeType, T, true, true> store(
            input_workspace.data<int8_t>(),
            dim_embed,
            ln_scale_data,
            ln_bias_data,
            qkv_in_scale[i + 1],
            quant_round_type,
            quant_max_bound,
            quant_min_bound);
        DispatchLayerNorm<decltype(load),
                          decltype(store),
                          LayerNormComputeType>(dev_ctx.stream(),
                                                load,
                                                store,
                                                token_num,
                                                dim_embed,
                                                epsilon,
                                                ln_mean_data,
                                                ln_var_data);
#ifdef DEBUG_MOE_TMPROFILE_INT8
        dev_ctx.Wait();
        ln_tm.Pause();
#endif
      } else {
        // last layer, only add residual, T
        phi::AddKernel<T, phi::GPUContext>(
            dev_ctx, all_gather_out, bias_dropout_residual_out, moe_out);
      }
      x_data = moe_out->data<T>();
#ifdef DEBUG_MOE_TMPROFILE_INT8
      dev_ctx.Wait();
      trans_tm.Pause();
#endif
    }  // end for layer loop
    if (encoder_remove_padding) {
      InvokeRebuildPadding(dev_ctx,
                           from_data,
                           moe_out->data<T>(),
                           padding_offset_data,
                           token_num,
                           dim_embed);
    }
    out->Resize({{bsz, seq_len, dim_embed}});
#ifdef DEBUG_MOE_TMPROFILE_INT8
    dev_ctx.Wait();
    all_tm.Pause();
    VLOG(0) << "gpu=" << static_cast<int>(dev_ctx.GetPlace().GetDeviceId())
            << ", bsz=" << bsz << ", seq_len=" << seq_len
            << ", total span=" << all_tm.ElapsedMS()
            << ", input=" << other_tm.ElapsedMS()
            << ", transformer=" << trans_tm.ElapsedMS()
            << ", [qkv=" << qkv_tm.ElapsedMS()
            << ", fmha=" << fmha_tm.ElapsedMS()
            << ", out_linear=" << out_linear_tm.ElapsedMS()
            << ", expert=" << expert_tm.ElapsedMS()
            << ", ln=" << ln_tm.ElapsedMS()
            << ", gate/all2all=" << gate_tm.ElapsedMS() << "/"
            << gate_nccl_tm.ElapsedMS()
            << ", scatter=" << scatter_tm.ElapsedMS()
            << ", gather=" << gather_tm.ElapsedMS() << "]";
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    fused_multi_transformer_moe_int8,
    ops::FusedMultiTransformerMoeINT8OpKernel<plat::float16>);